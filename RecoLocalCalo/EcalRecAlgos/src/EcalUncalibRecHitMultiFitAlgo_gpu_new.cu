#include "hip/hip_runtime.h"
#include "RecoLocalCalo/EcalRecAlgos/interface/EcalUncalibRecHitMultiFitAlgo_gpu_new.h"

#include "DataFormats/EcalDigi/interface/EcalDigiCollections.h"
#include "CondFormats/EcalObjects/interface/EcalPedestals.h"
#include "CondFormats/EcalObjects/interface/EcalMGPAGainRatio.h"
#include "CondFormats/EcalObjects/interface/EcalXtalGroupId.h"
#include "CondFormats/EcalObjects/interface/EcalPulseShapes.h"
#include "CondFormats/EcalObjects/interface/EcalPulseCovariances.h"
#include "CondFormats/EcalObjects/interface/EcalSampleMask.h"

#include <iostream>

#include "DataFormats/EcalDigi/interface/EcalDataFrame.h"
#include "RecoLocalCalo/EcalRecAlgos/interface/Common.h"

#include "RecoLocalCalo/EcalRecAlgos/interface/inplace_fnnls.h"

#include "hip/hip_runtime.h"

//#define DEBUG

namespace ecal { namespace multifit { namespace v1 {

///
/// assume kernel launch configuration is 
/// (MAXSAMPLES * nchannels, blocks)
/// 
__global__
void kernel_prep_1d(EcalPulseShape const* shapes_in,
                    FullSampleVector* shapes_out, 
                    uint16_t const* digis_in,
                    SampleVector* amplitudes,
                    SampleGainVector* gainsNoise,
                    SampleGainVector* gainsPedestal,
                    float const* mean_x1,
                    float const* mean_x12,
                    float const* mean_x6,
                    float const* gain6Over1,
                    float const* gain12Over6,
                    int nchannels) {
    constexpr bool dynamicPedestal = false;
    constexpr int nsamples = EcalDataFrame::MAXSAMPLES;
    int tx = threadIdx.x + blockIdx.x*blockDim.x;
    int ch = tx / nsamples;
    if (ch < nchannels) {
        //
        // pulse shape template
        //
        int sample = threadIdx.x % nsamples;
        for (int isample=sample; isample<EcalPulseShape::TEMPLATESAMPLES; 
            isample+=nsamples)
            shapes_out[ch](isample + 7) = shapes_in[ch].pdfval[isample];

        //
        // amplitudes
        //
        int adc = ecalMGPA::adc(digis_in[tx]);
        int gainId = ecalMGPA::gainId(digis_in[tx]);
        float amplitude = 0.f;
        float pedestal = 0.f;
        float gainratio = 0.f;

        // TODO: divergent branch
        if (gainId==0 || gainId==3) {
            pedestal = mean_x1[ch];
            gainratio = gain6Over1[ch] * gain12Over6[ch];
            gainsNoise[ch](sample) = 2;
            gainsPedestal[ch](sample) = dynamicPedestal ? 2 : -1;
        } else if (gainId==1) {
            pedestal = mean_x12[ch];
            gainratio = 1.;
            gainsNoise[ch](sample) = 0;
            gainsPedestal[ch](sample) = dynamicPedestal ? 0 : -1;
        } else if (gainId==2) {
            pedestal = mean_x6[ch];
            gainratio = gain12Over6[ch];
            gainsNoise[ch](sample)  = 1;
            gainsPedestal[ch](sample) = dynamicPedestal ? 1 : -1;
        }

        // TODO: compile time constant -> branch should be non-divergent
        if (dynamicPedestal)
            amplitude = static_cast<float>(adc) * gainratio;
        else
            amplitude = static_cast<float>(adc - pedestal) * gainratio;

        amplitudes[ch][sample] = amplitude;
    }
}

///
/// assume kernel launch configuration is 
/// ([MAXSAMPLES, MAXSAMPLES], nchannels)
///
__global__
void kernel_prep_2d(EcalPulseCovariance const* pulse_cov_in,
                    FullSampleMatrix* pulse_cov_out,
                    SampleGainVector const* gainNoise,
                    SampleMatrixD const* noisecorrs,
                    float const* rms_x12,
                    float const* rms_x6,
                    float const* rms_x1,
                    float const* gain12Over6,
                    float const* gain6Over1,
                    SampleMatrix* noisecov,
                    PulseMatrixType* pulse_matrix,
                    FullSampleVector const* pulse_shape,
                    BXVectorType const* bxs) {
    int ch = blockIdx.x;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    constexpr int nsamples = EcalDataFrame::MAXSAMPLES;
    constexpr float addPedestalUncertainty = 0.f;
    constexpr bool dynamicPedestal = false;
    constexpr int template_samples = EcalPulseShape::TEMPLATESAMPLES;

    for (int iy=ty, ix=tx; ix<=template_samples && iy<=template_samples; 
        ix+=nsamples, iy+=nsamples)
        pulse_cov_out[ch](iy+7, ix+7) = pulse_cov_in[ch].covval[iy][ix];
    
    bool hasGainSwitch = false;
    // non-divergent branch
    if (hasGainSwitch) {
        // TODO: did not include simplified noise model
        float noise_value = 0;
        // 
        int gainidx=0;
        char mask = gainidx;
        int pedestal = gainNoise[ch][ty] == mask ? 1 : 0;
        noise_value += /* gainratio is 1*/ rms_x12[ch]*rms_x12[ch]
            *pedestal*noisecorrs[0](ty, tx);
        // non-divergent branch
        if (!dynamicPedestal && addPedestalUncertainty>0.f) {
            noise_value += /* gainratio is 1 */
                addPedestalUncertainty*addPedestalUncertainty*pedestal;
        }

        //
        gainidx=1;
        mask = gainidx;
        pedestal = gainNoise[ch][ty] == mask ? 1 : 0;
        noise_value += gain12Over6[ch]*gain12Over6[ch]
            *rms_x6[ch]*rms_x6[ch]*pedestal*noisecorrs[1](ty, tx);
        // non-divergent branch
        if (!dynamicPedestal && addPedestalUncertainty>0.f) {
            noise_value += gain12Over6[ch]*gain12Over6[ch]
                *addPedestalUncertainty*addPedestalUncertainty
                *pedestal;
        }
        
        //
        gainidx=2;
        mask = gainidx;
        pedestal = gainNoise[ch][ty] == mask ? 1 : 0;
        float tmp = gain6Over1[ch] * gain12Over6[ch];
        noise_value += tmp*tmp * rms_x1[ch]*rms_x1[ch]
            *pedestal*noisecorrs[2](ty, tx);
        // non-divergent branch
        if (!dynamicPedestal && addPedestalUncertainty>0.f) {
            noise_value += tmp*tmp * addPedestalUncertainty*addPedestalUncertainty
                * pedestal;
        }

        noisecov[ch](ty, tx) = noise_value;
    } else {
        auto rms = rms_x12[ch];
        float noise_value = rms*rms * noisecorrs[0](ty, tx);
        if (!dynamicPedestal && addPedestalUncertainty>0.f)
            noise_value += addPedestalUncertainty*addPedestalUncertainty;
        noisecov[ch](ty, tx) = noise_value;
    }

    // pulse matrix
    int bx = (*bxs)(tx);
    int offset = 7 - 3 - bx;
    float value = pulse_shape[ch](offset + ty);
    pulse_matrix[ch](ty, tx) = value;
}

__device__
bool update_covariance(SampleMatrix const& noisecov,
                       FullSampleMatrix const& full_pulse_cov,
                       SampleMatrix& inverse_cov,
                       BXVectorType const& bxs,
                       SampleDecompLLT& covariance_decomposition,
                       SampleVector const& amplitudes) {
    constexpr int nsamples = SampleVector::RowsAtCompileTime;
    constexpr int npulses = BXVectorType::RowsAtCompileTime;

    inverse_cov = noisecov;

    for (unsigned int ipulse=0; ipulse<npulses; ipulse++) {
        if (amplitudes.coeff(ipulse) == 0) 
            continue;

        int bx = bxs.coeff(ipulse);
        int first_sample_t = std::max(0, bx+3);
        int offset = 7 - 3 - bx;

        float value = amplitudes.coeff(ipulse);
        float value_sq = value*value;

        unsigned int nsample_pulse = nsamples - first_sample_t;
        inverse_cov.block(first_sample_t, first_sample_t, 
                          nsample_pulse, nsample_pulse)
            += value_sq * full_pulse_cov.block(first_sample_t + offset,
                                               first_sample_t + offset,
                                               nsample_pulse,
                                               nsample_pulse);
    }

    covariance_decomposition.compute(inverse_cov);
    return true;
}

__device__
float compute_chi2(SampleDecompLLT& covariance_decomposition,
                   PulseMatrixType const& pulse_matrix,
                   SampleVector const& amplitudes,
                   SampleVector const& samples) {
    return covariance_decomposition.matrixL()
        .solve(pulse_matrix * amplitudes - samples)
        .squaredNorm();
}

///
/// launch ctx parameters are (nchannels / block, blocks)
/// TODO: trivial impl for now, there must be a way to improve
///
/// Conventions:
///   - amplitudes -> solution vector, what we are fitting for
///   - samples -> raw detector responses
///   - passive constraint - satisfied constraint
///   - active constraint - unsatisfied (yet) constraint
///
__global__
void kernel_minimize(SampleMatrix const* noisecov,
                     FullSampleMatrix const* full_pulse_cov,
                     BXVectorType const* bxs,
                     SampleVector const* samples,
                     SampleVector* amplitudes,
                     float* energies,
                     PulseMatrixType* pulse_matrix, 
                     bool* statuses,
                     float* chi2s,
                     int nchannels,
                     int max_iterations) {
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    if (idx < nchannels) {
        bool status = false;
        int iter = 0;
        SampleDecompLLT covariance_decomposition;
        SampleMatrix inverse_cov;
        int npassive = 0;
        amplitudes[idx] = SampleVector::Zero();
        float chi2 = 0;
        while (true) {
            if (iter >= max_iterations)
                break;

            // TODO
            status = update_covariance(
                noisecov[idx], 
                full_pulse_cov[idx],
                inverse_cov,
                *bxs,
                covariance_decomposition,
                amplitudes[idx]);
            if (!status) 
                break;

            // TODO
            SampleMatrix A = covariance_decomposition.matrixL()
                .solve(pulse_matrix[idx]);
            SampleVector b = covariance_decomposition.matrixL()
                .solve(samples[idx]);
            
            status = inplace_fnnls(
                A, b, amplitudes[idx],
                npassive);
                
            if (!status)
                break;

            // TODO
            float chi2_now = compute_chi2(
                covariance_decomposition,
                pulse_matrix[idx],
                amplitudes[idx],
                samples[idx]);
            float deltachi2 = chi2_now - chi2;
            if (ecal::abs(deltachi2) < 1e-3)
                break;

            chi2 = chi2_now;
            ++iter;
        }

        float energy = amplitudes[idx](5);
        energies[idx] = energy; // according to bxs vector bxs[5] = 0
        statuses[idx] = status;
        chi2s[idx] = chi2;
    }
}

///
/// Build an Ecal RecHit.
/// TODO: Use SoA data structures on the host directly
/// the reason for removing this from minimize kernel is to isolate the minimize + 
/// again, building an aos rec hit involves strides... -> bad memory access pattern
///
__global__
void kernel_build_rechit(
    float const* energies,
    float const* chi2s,
    uint32_t* dids,
    EcalUncalibratedRecHit* rechits,
    int nchannels) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < nchannels) {
        rechits[idx] = EcalUncalibratedRecHit{dids[idx], energies[idx],
            0, 0, chi2s[idx], 0};
    }
}

void scatter(host_data& h_data, device_data& d_data, conf_data const& conf) {

/*
void scatter(EcalDigiCollection const& digis,
             EcalUncalibratedRecHitCollection& rechits,
             std::vector<EcalPedestal> const& vpedestals,
             std::vector<EcalMGPAGainRatio> const& vgains,
             std::vector<EcalXtalGroupId> const& vxtals,
             std::vector<EcalPulseShape> const& vpulses,
             std::vector<EcalPulseCovariance> const& vcovariances,
             SampleMatrixGainArray const& noisecors,
             device_data &d_data) {
*/
    auto const& ids = h_data.digis->ids();
    auto const& digis_data = h_data.digis->data();
    using digis_type = std::vector<uint16_t>;
    using dids_type = std::vector<uint32_t>;
    
    //
    // TODO: remove per event alloc/dealloc -> do once at the start
    //
    /*
    hipMalloc((void**)&d_digis_data,
        digis_data.size() * sizeof(digis_type::value_type));
    hipMalloc((void**)&d_ids,
        ids.size() * sizeof(dids_type::value_type));
    hipMalloc((void**)&d_pedestals,
        vpedestals.size() * sizeof(EcalPedestal));
    hipMalloc((void**)&d_gains, 
        vgains.size() * sizeof(EcalMGPAGainRatio));
    hipMalloc((void**)&d_xtals,
        vxtals.size() * sizeof(EcalXtalGroupId));
    hipMalloc((void**)&d_shapes,
        vpulses.size() * sizeof(EcalPulseShape));
    hipMalloc((void**)&d_covariances,
        vcovariances.size() * sizeof(EcalPulseCovariance));
    hipMalloc((void**)&d_rechits,
        rechits.size() * sizeof(EcalUncalibratedRecHit));
    hipMalloc((void**)&d_noisecors,
        noisecors.size() * sizeof(SampleMatrix));
    ecal::cuda::assert_if_error();
    */

    // 
    // copy to the device
    // TODO: can conditions be copied only once when updated?
    //
    hipMemcpy(d_data.digis_data, digis_data.data(),
        digis_data.size() * sizeof(digis_type::value_type),
        hipMemcpyHostToDevice);
    hipMemcpy(d_data.ids, ids.data(),
        ids.size() * sizeof(dids_type::value_type),
        hipMemcpyHostToDevice);
 
//    hipMemcpy(d_data.pedestals, h_data.pedestals->data(),
//        h_data.pedestals->size() * sizeof(EcalPedestal),
//        hipMemcpyHostToDevice);

    hipMemcpy(d_data.mean_x12, h_data.ped_data.mean_x12.data(),
        h_data.ped_data.mean_x12.size() * sizeof(float),
        hipMemcpyHostToDevice);
    hipMemcpy(d_data.rms_x12, h_data.ped_data.rms_x12.data(),
        h_data.ped_data.rms_x12.size() * sizeof(float),
        hipMemcpyHostToDevice);
    hipMemcpy(d_data.mean_x6, h_data.ped_data.mean_x6.data(),
        h_data.ped_data.mean_x6.size() * sizeof(float),
        hipMemcpyHostToDevice);
    hipMemcpy(d_data.rms_x6, h_data.ped_data.rms_x6.data(),
        h_data.ped_data.rms_x6.size() * sizeof(float),
        hipMemcpyHostToDevice);
    hipMemcpy(d_data.mean_x1, h_data.ped_data.mean_x1.data(),
        h_data.ped_data.mean_x1.size() * sizeof(float),
        hipMemcpyHostToDevice);
    hipMemcpy(d_data.rms_x1, h_data.ped_data.rms_x1.data(),
        h_data.ped_data.rms_x1.size() * sizeof(float),
        hipMemcpyHostToDevice);

//    hipMemcpy(d_data.gains, h_data.gains->data(),
//        h_data.gains->size() * sizeof(EcalMGPAGainRatio),
//        hipMemcpyHostToDevice);

    hipMemcpy(d_data.gain12Over6, h_data.gainratio_data.gain12Over6.data(),
        h_data.gainratio_data.gain12Over6.size() * sizeof(float),
        hipMemcpyHostToDevice);
    hipMemcpy(d_data.gain6Over1, h_data.gainratio_data.gain6Over1.data(),
        h_data.gainratio_data.gain6Over1.size() * sizeof(float),
        hipMemcpyHostToDevice);

    hipMemcpy(d_data.xtals, h_data.xtals->data(),
        h_data.xtals->size() * sizeof(EcalXtalGroupId),
        hipMemcpyHostToDevice);
    hipMemcpy(d_data.pulses, h_data.pulse_shapes->data(),
        h_data.pulse_shapes->size() * sizeof(EcalPulseShape),
        hipMemcpyHostToDevice);
    hipMemcpy(d_data.covariances, h_data.pulse_covariances->data(),
        h_data.pulse_covariances->size() * sizeof(EcalPulseCovariance),
        hipMemcpyHostToDevice);

    hipMemcpy(d_data.noisecorrs, h_data.noisecorrs->data(),
        h_data.noisecorrs->size() * sizeof(SampleMatrixD),
        hipMemcpyHostToDevice);

    hipMemcpy(d_data.sample_mask, h_data.sample_mask,
        sizeof(EcalSampleMask),
        hipMemcpyHostToDevice);
    hipMemcpy(d_data.EBTimeCorrAmplitudeBins, 
        h_data.time_bias_corrections->EBTimeCorrAmplitudeBins.data(),
        sizeof(float) * h_data.time_bias_corrections->EBTimeCorrAmplitudeBins.size(),
        hipMemcpyHostToDevice);
    hipMemcpy(d_data.EBTimeCorrShiftBins, 
        h_data.time_bias_corrections->EBTimeCorrShiftBins.data(),
        sizeof(float) * h_data.time_bias_corrections->EBTimeCorrShiftBins.size(),
        hipMemcpyHostToDevice);
    hipMemcpy(d_data.EETimeCorrAmplitudeBins, 
        h_data.time_bias_corrections->EETimeCorrAmplitudeBins.data(),
        sizeof(float) * h_data.time_bias_corrections->EETimeCorrAmplitudeBins.size(),
        hipMemcpyHostToDevice);
    hipMemcpy(d_data.EETimeCorrShiftBins, 
        h_data.time_bias_corrections->EETimeCorrShiftBins.data(),
        sizeof(float) * h_data.time_bias_corrections->EETimeCorrShiftBins.size(),
        hipMemcpyHostToDevice);
    hipMemcpy(d_data.weights,
        h_data.weights->data(),
        sizeof(EMatrix) * h_data.weights->size(),
        hipMemcpyHostToDevice);
    hipMemcpy(d_data.bxs, h_data.bxs,
        sizeof(BXVectorType),
        hipMemcpyHostToDevice);
    ecal::cuda::assert_if_error();

    int nthreads_per_block = conf.threads.x;
    int nblocks = (h_data.digis->size() + nthreads_per_block - 1) / nthreads_per_block;

    std::cout << "new impl running\n";
    
    // 
    // 1d preparation kernel
    //
    unsigned int nchannels_per_block = 32;
    unsigned int threads_1d = 10 * nchannels_per_block;
    unsigned int blocks_1d = threads_1d > 10*h_data.digis->size() 
        ? 1 : (h_data.digis->size() + threads_1d - 1) / threads_1d;
    kernel_prep_1d<<<blocks_1d, threads_1d>>>(
        d_data.pulses, d_data.epulses,
        d_data.digis_data, d_data.samples,
        d_data.gainsNoise,
        d_data.gainsPedestal,
        d_data.mean_x1,
        d_data.mean_x12,
        d_data.mean_x6,
        d_data.gain6Over1,
        d_data.gain12Over6,
        h_data.digis->size());
    hipDeviceSynchronize();
    ecal::cuda::assert_if_error();

    //
    // 2d preparation kernel
    //
    int blocks_2d = h_data.digis->size();
    dim3 threads_2d{10, 10};
    kernel_prep_2d<<<blocks_2d, threads_2d>>>(
        d_data.covariances, d_data.pulse_covariances,
        d_data.gainsNoise,
        d_data.noisecorrs,
        d_data.rms_x12,
        d_data.rms_x6,
        d_data.rms_x1,
        d_data.gain12Over6,
        d_data.gain6Over1,
        d_data.noisecov,
        d_data.pulse_matrix,
        d_data.epulses,
        d_data.bxs);
    hipDeviceSynchronize();
    ecal::cuda::assert_if_error();
//    kernel_minimize<<<>>>();

    unsigned int threads_min = conf.threads.x;
    unsigned int blocks_min = threads_min > h_data.digis->size()
        ? 1 : (h_data.digis->size() + threads_min - 1) / threads_min;
    kernel_minimize<<<blocks_min, threads_min>>>(
        d_data.noisecov,
        d_data.pulse_covariances,
        d_data.bxs,
        d_data.samples,
        d_data.amplitudes,
        d_data.energies,
        d_data.pulse_matrix,
        d_data.statuses,
        d_data.chi2,
        h_data.digis->size(),
        50);
    hipDeviceSynchronize();
    ecal::cuda::assert_if_error();
/*
    kernel_build_rechit<<<blocks_min, threads_min>>>(
        d_data.energies,
        d_data.chi2,
        d_data.ids,
        d_data.rechits,
        h_data.digis->size());
    hipDeviceSynchronize();
    ecal::cuda::assert_if_error();
    */

  /*  kernel_reconstruct<<<nblocks, nthreads_per_block>>>(
        d_data.digis_data,
        d_data.ids,*/
        /* d_rechits, */
/*        d_data.pedestals,
        d_data.gains,
        d_data.xtals,
        d_data.pulses,
        d_data.covariances,
        d_data.rechits,
        d_data.noisecors,
        d_data.sample_mask,
        d_data.EBTimeCorrAmplitudeBins, 
        h_data.time_bias_corrections->EBTimeCorrAmplitudeBins.size(),
        d_data.EBTimeCorrShiftBins, 
        h_data.time_bias_corrections->EBTimeCorrShiftBins.size(),
        d_data.EETimeCorrAmplitudeBins, 
        h_data.time_bias_corrections->EETimeCorrAmplitudeBins.size(),
        d_data.EETimeCorrShiftBins, 
        h_data.time_bias_corrections->EETimeCorrShiftBins.size(),
        d_data.weights,
        h_data.digis->size()
    );*/

    //
    // transfer the results back
    //
    hipMemcpy(&(*h_data.rechits_soa.amplitude.begin()),
               d_data.energies,
               h_data.rechits_soa.amplitude.size() * sizeof(float),
               hipMemcpyDeviceToHost);
    hipMemcpy(&(*h_data.rechits_soa.chi2.begin()),
               d_data.chi2,
               h_data.rechits_soa.chi2.size() * sizeof(float),
               hipMemcpyDeviceToHost);
    hipMemcpy(&(*h_data.rechits_soa.did.begin()),
               d_data.ids,
               h_data.rechits_soa.did.size() * sizeof(uint32_t),
               hipMemcpyDeviceToHost);

//    hipMemcpy(&(*h_data.rechits->begin()), d_data.rechits,
//        h_data.rechits->size() * sizeof(EcalUncalibratedRecHit),
//        hipMemcpyDeviceToHost);

    // 
    // free all the device ptrs
    // TODO: remove per event dealloc
    //
    /*
    hipFree(d_digis_data);
    hipFree(d_ids);
    hipFree(d_pedestals);
    hipFree(d_gains);
    hipFree(d_xtals);
    hipFree(d_shapes);
    hipFree(d_covariances);
    hipFree(d_rechits);
    hipFree(d_noisecors);
    ecal::cuda::assert_if_error();
    */
}

}}}

/*
#include "FWCore/MessageLogger/interface/MessageLogger.h"

#include "CondFormats/EcalObjects/interface/EcalPedestals.h"
#include "CondFormats/EcalObjects/interface/EcalGainRatios.h"

EcalUncalibRecHitMultiFitAlgo::EcalUncalibRecHitMultiFitAlgo() : 
  _computeErrors(true),
  _doPrefit(false),
  _prefitMaxChiSq(1.0),
  _dynamicPedestals(false),
  _mitigateBadSamples(false),
  _selectiveBadSampleCriteria(false),
  _addPedestalUncertainty(0.),
  _simplifiedNoiseModelForGainSwitch(true),
  _gainSwitchUseMaxSample(false){
    
  _singlebx.resize(1);
  _singlebx << 0;
  
  _pulsefuncSingle.disableErrorCalculation();
  _pulsefuncSingle.setMaxIters(1);
  _pulsefuncSingle.setMaxIterWarnings(false);
    
}

/// compute rechits
EcalUncalibratedRecHit EcalUncalibRecHitMultiFitAlgo::makeRecHit(const EcalDataFrame& dataFrame, const EcalPedestals::Item * aped, const EcalMGPAGainRatio * aGain, const SampleMatrixGainArray &noisecors, const FullSampleVector &fullpulse, const FullSampleMatrix &fullpulsecov, const BXVector &activeBX) {

  uint32_t flags = 0;
  
  const unsigned int nsample = EcalDataFrame::MAXSAMPLES;
  
  double maxamplitude = -std::numeric_limits<double>::max();
  const unsigned int iSampleMax = 5;
  const unsigned int iFullPulseMax = 9;
  
  double pedval = 0.;
    
  SampleVector amplitudes;
  SampleGainVector gainsNoise;
  SampleGainVector gainsPedestal;
  SampleGainVector badSamples = SampleGainVector::Zero();
  bool hasSaturation = dataFrame.isSaturated();
  bool hasGainSwitch = hasSaturation || dataFrame.hasSwitchToGain6() || dataFrame.hasSwitchToGain1();
  
  //no dynamic pedestal in case of gain switch, since then the fit becomes too underconstrained
  bool dynamicPedestal = _dynamicPedestals && !hasGainSwitch;
  
  for(unsigned int iSample = 0; iSample < nsample; iSample++) {
        
    const EcalMGPASample &sample = dataFrame.sample(iSample);
    
    double amplitude = 0.;
    int gainId = sample.gainId();
    
    double pedestal = 0.;
    double gainratio = 1.;
    
    if (gainId==0 || gainId==3) {
      pedestal = aped->mean_x1;
      gainratio = aGain->gain6Over1()*aGain->gain12Over6();
      gainsNoise[iSample] = 2;
      gainsPedestal[iSample] = dynamicPedestal ? 2 : -1;  //-1 for static pedestal
    }
    else if (gainId==1) {
      pedestal = aped->mean_x12;
      gainratio = 1.;
      gainsNoise[iSample] = 0;
      gainsPedestal[iSample] = dynamicPedestal ? 0 : -1; //-1 for static pedestal
    }
    else if (gainId==2) {
      pedestal = aped->mean_x6;
      gainratio = aGain->gain12Over6();
      gainsNoise[iSample] = 1;
      gainsPedestal[iSample] = dynamicPedestal ? 1 : -1; //-1 for static pedestals
    }

    if (dynamicPedestal) {
      amplitude = (double)(sample.adc())*gainratio;
    }
    else {
      amplitude = ((double)(sample.adc()) - pedestal) * gainratio;
    }
    
    if (gainId == 0) {
       edm::LogError("EcalUncalibRecHitMultiFitAlgo")<< "Saturation encountered.  Multifit is not intended to be used for saturated channels.";
      //saturation
      if (dynamicPedestal) {
        amplitude = 4095.*gainratio;
      }
      else {
        amplitude = (4095. - pedestal) * gainratio;
      }
    }
        
    amplitudes[iSample] = amplitude;
    
    if (iSample==iSampleMax) {
      maxamplitude = amplitude;
      pedval = pedestal;
    }
        
  }

  double amplitude, amperr, chisq;
  bool status = false;
    
  //special handling for gain switch, where sample before maximum is potentially affected by slew rate limitation
  //optionally apply a stricter criteria, assuming slew rate limit is only reached in case where maximum sample has gain switched but previous sample has not
  //option 1: use simple max-sample algorithm
  if (hasGainSwitch && _gainSwitchUseMaxSample) {
    double maxpulseamplitude = maxamplitude / fullpulse[iFullPulseMax];
    EcalUncalibratedRecHit rh( dataFrame.id(), maxpulseamplitude, pedval, 0., 0., flags );
    rh.setAmplitudeError(0.);
    for (unsigned int ipulse=0; ipulse<_pulsefunc.BXs().rows(); ++ipulse) {
      int bx = _pulsefunc.BXs().coeff(ipulse);
      if (bx!=0) {
        rh.setOutOfTimeAmplitude(bx+5, 0.0);
      }
    }
    return rh;
  }

  //option2: A floating negative single-sample offset is added to the fit
  //such that the affected sample is treated only as a lower limit for the true amplitude
  bool mitigateBadSample = _mitigateBadSamples && hasGainSwitch && iSampleMax>0;
  mitigateBadSample &= (!_selectiveBadSampleCriteria || (gainsNoise.coeff(iSampleMax-1)!=gainsNoise.coeff(iSampleMax)) );
  if (mitigateBadSample) {
    badSamples[iSampleMax-1] = 1;
  }
  
  //compute noise covariance matrix, which depends on the sample gains
  SampleMatrix noisecov;
  if (hasGainSwitch) {
    std::array<double,3> pedrmss = {{aped->rms_x12, aped->rms_x6, aped->rms_x1}};
    std::array<double,3> gainratios = {{ 1., aGain->gain12Over6(), aGain->gain6Over1()*aGain->gain12Over6()}};
    if (_simplifiedNoiseModelForGainSwitch) {
      int gainidxmax = gainsNoise[iSampleMax];
      noisecov = gainratios[gainidxmax]*gainratios[gainidxmax]*pedrmss[gainidxmax]*pedrmss[gainidxmax]*noisecors[gainidxmax];
      if (!dynamicPedestal && _addPedestalUncertainty>0.) {
        //add fully correlated component to noise covariance to inflate pedestal uncertainty
        noisecov += _addPedestalUncertainty*_addPedestalUncertainty*SampleMatrix::Ones();
      }
    }
    else {
      noisecov = SampleMatrix::Zero();
      for (unsigned int gainidx=0; gainidx<noisecors.size(); ++gainidx) {
        SampleGainVector mask = gainidx*SampleGainVector::Ones();
        SampleVector pedestal = (gainsNoise.array()==mask.array()).cast<SampleVector::value_type>();
        if (pedestal.maxCoeff()>0.) {
          //select out relevant components of each correlation matrix, and assume no correlation between samples with
          //different gain
          noisecov += gainratios[gainidx]*gainratios[gainidx]*pedrmss[gainidx]*pedrmss[gainidx]*pedestal.asDiagonal()*noisecors[gainidx]*pedestal.asDiagonal();
          if (!dynamicPedestal && _addPedestalUncertainty>0.) {
            //add fully correlated component to noise covariance to inflate pedestal uncertainty
            noisecov += gainratios[gainidx]*gainratios[gainidx]*_addPedestalUncertainty*_addPedestalUncertainty*pedestal.asDiagonal()*SampleMatrix::Ones()*pedestal.asDiagonal();
          }
        }
      }
    }
  }
  else {
    noisecov = aped->rms_x12*aped->rms_x12*noisecors[0];
    if (!dynamicPedestal && _addPedestalUncertainty>0.) {
      //add fully correlated component to noise covariance to inflate pedestal uncertainty
      noisecov += _addPedestalUncertainty*_addPedestalUncertainty*SampleMatrix::Ones();
    }
  }
  
  //optimized one-pulse fit for hlt
  bool usePrefit = false;
  if (_doPrefit) {
    status = _pulsefuncSingle.DoFit(amplitudes,noisecov,_singlebx,fullpulse,fullpulsecov,gainsPedestal,badSamples);
    amplitude = status ? _pulsefuncSingle.X()[0] : 0.;
    amperr = status ? _pulsefuncSingle.Errors()[0] : 0.;
    chisq = _pulsefuncSingle.ChiSq();
    
    if (chisq < _prefitMaxChiSq) {
      usePrefit = true;
    }
  }
  
  if (!usePrefit) {
  
    if(!_computeErrors) _pulsefunc.disableErrorCalculation();
    status = _pulsefunc.DoFit(amplitudes,noisecov,activeBX,fullpulse,fullpulsecov,gainsPedestal,badSamples);
    chisq = _pulsefunc.ChiSq();
    
    if (!status) {
      edm::LogWarning("EcalUncalibRecHitMultiFitAlgo::makeRecHit") << "Failed Fit" << std::endl;
    }

    unsigned int ipulseintime = 0;
    for (unsigned int ipulse=0; ipulse<_pulsefunc.BXs().rows(); ++ipulse) {
      if (_pulsefunc.BXs().coeff(ipulse)==0) {
        ipulseintime = ipulse;
        break;
      }
    }
    
    amplitude = status ? _pulsefunc.X()[ipulseintime] : 0.;
    amperr = status ? _pulsefunc.Errors()[ipulseintime] : 0.;
  
  }
  
  double jitter = 0.;
  
  EcalUncalibratedRecHit rh( dataFrame.id(), amplitude , pedval, jitter, chisq, flags );
  rh.setAmplitudeError(amperr);
  
  if (!usePrefit) {
    for (unsigned int ipulse=0; ipulse<_pulsefunc.BXs().rows(); ++ipulse) {
      int bx = _pulsefunc.BXs().coeff(ipulse);
      if (bx!=0 && std::abs(bx)<100) {
        rh.setOutOfTimeAmplitude(bx+5, status ? _pulsefunc.X().coeff(ipulse) : 0.);
      }
      else if (bx==(100+gainsPedestal[iSampleMax])) {
        rh.setPedestal(status ? _pulsefunc.X().coeff(ipulse) : 0.);
      }
    }
  }
  
  return rh;
}
*/
