#include "RecoLocalCalo/EcalRecAlgos/interface/EcalUncalibRecHitMultiFitAlgo_gpu_new.h"

#include "DataFormats/EcalDigi/interface/EcalDigiCollections.h"
#include "CondFormats/EcalObjects/interface/EcalPedestals.h"
#include "CondFormats/EcalObjects/interface/EcalMGPAGainRatio.h"
#include "CondFormats/EcalObjects/interface/EcalXtalGroupId.h"
#include "CondFormats/EcalObjects/interface/EcalPulseShapes.h"
#include "CondFormats/EcalObjects/interface/EcalPulseCovariances.h"
#include "CondFormats/EcalObjects/interface/EcalSampleMask.h"
#include "CondFormats/EcalObjects/interface/EcalSamplesCorrelation.h"

#include <iostream>
#include <limits>

#include "DataFormats/EcalDigi/interface/EcalDataFrame.h"
#include "RecoLocalCalo/EcalRecAlgos/interface/Common.h"

#include "hip/hip_runtime.h"

#include "AmplitudeComputationCommonKernels.h"
#include "AmplitudeComputationKernelsV1.h"
#include "AmplitudeComputationKernelsV2.h"
#include "TimeComputationKernels.h"

//#define DEBUG

//#define ECAL_RECO_CUDA_DEBUG

namespace ecal { namespace multifit {
    
void scatter(host_data& h_data, device_data& d_data, conf_data const& conf) {
    auto const& ids = h_data.digis->ids();
    auto const& digis_data = h_data.digis->data();
    using digis_type = std::vector<uint16_t>;
    using dids_type = std::vector<uint32_t>;
    bool barrel = 
        DetId{h_data.digis->begin()->id()}
            .subdetId() == EcalBarrel;
    bool gainSwitchUseMaxSample = barrel; // accodring to the cpu setup

    // 
    // copy to the device
    // TODO: can conditions be copied only once when updated?
    //
    hipMemcpy(d_data.digis_data, digis_data.data(),
        digis_data.size() * sizeof(digis_type::value_type),
        hipMemcpyHostToDevice);
    hipMemcpy(d_data.ids, ids.data(),
        ids.size() * sizeof(dids_type::value_type),
        hipMemcpyHostToDevice);
 
    hipMemcpy(d_data.mean_x12, h_data.ped_data.mean_x12.data(),
        h_data.ped_data.mean_x12.size() * sizeof(float),
        hipMemcpyHostToDevice);
    hipMemcpy(d_data.rms_x12, h_data.ped_data.rms_x12.data(),
        h_data.ped_data.rms_x12.size() * sizeof(float),
        hipMemcpyHostToDevice);
    hipMemcpy(d_data.mean_x6, h_data.ped_data.mean_x6.data(),
        h_data.ped_data.mean_x6.size() * sizeof(float),
        hipMemcpyHostToDevice);
    hipMemcpy(d_data.rms_x6, h_data.ped_data.rms_x6.data(),
        h_data.ped_data.rms_x6.size() * sizeof(float),
        hipMemcpyHostToDevice);
    hipMemcpy(d_data.mean_x1, h_data.ped_data.mean_x1.data(),
        h_data.ped_data.mean_x1.size() * sizeof(float),
        hipMemcpyHostToDevice);
    hipMemcpy(d_data.rms_x1, h_data.ped_data.rms_x1.data(),
        h_data.ped_data.rms_x1.size() * sizeof(float),
        hipMemcpyHostToDevice);

    hipMemcpy(d_data.gain12Over6, h_data.gainratio_data.gain12Over6.data(),
        h_data.gainratio_data.gain12Over6.size() * sizeof(float),
        hipMemcpyHostToDevice);
    hipMemcpy(d_data.gain6Over1, h_data.gainratio_data.gain6Over1.data(),
        h_data.gainratio_data.gain6Over1.size() * sizeof(float),
        hipMemcpyHostToDevice);

    hipMemcpy(d_data.pulses, h_data.pulse_shapes->data(),
        h_data.pulse_shapes->size() * sizeof(EcalPulseShape),
        hipMemcpyHostToDevice);
    hipMemcpy(d_data.covariances, h_data.pulse_covariances->data(),
        h_data.pulse_covariances->size() * sizeof(EcalPulseCovariance),
        hipMemcpyHostToDevice);

    hipMemcpy(d_data.G12SamplesCorrelation, 
               barrel
                 ? h_data.noiseCovariances->EBG12SamplesCorrelation.data()
                 : h_data.noiseCovariances->EEG12SamplesCorrelation.data(),
               EcalDataFrame::MAXSAMPLES * sizeof(double),
               hipMemcpyHostToDevice);
    hipMemcpy(d_data.G6SamplesCorrelation, 
               barrel
                 ? h_data.noiseCovariances->EBG6SamplesCorrelation.data()
                 : h_data.noiseCovariances->EEG6SamplesCorrelation.data(),
               EcalDataFrame::MAXSAMPLES * sizeof(double),
               hipMemcpyHostToDevice);
    hipMemcpy(d_data.G1SamplesCorrelation, 
               barrel
                 ? h_data.noiseCovariances->EBG1SamplesCorrelation.data()
                 : h_data.noiseCovariances->EEG1SamplesCorrelation.data(),
               EcalDataFrame::MAXSAMPLES * sizeof(double),
               hipMemcpyHostToDevice);

    if (barrel) {
        hipMemcpy(d_data.EBTimeCorrAmplitudeBins, 
            h_data.time_bias_corrections->EBTimeCorrAmplitudeBins.data(),
            sizeof(float) * h_data.time_bias_corrections->EBTimeCorrAmplitudeBins.size(),
            hipMemcpyHostToDevice);
        hipMemcpy(d_data.EBTimeCorrShiftBins, 
            h_data.time_bias_corrections->EBTimeCorrShiftBins.data(),
            sizeof(float) * h_data.time_bias_corrections->EBTimeCorrShiftBins.size(),
            hipMemcpyHostToDevice);
    } else {
        hipMemcpy(d_data.EETimeCorrAmplitudeBins, 
            h_data.time_bias_corrections->EETimeCorrAmplitudeBins.data(),
            sizeof(float) * h_data.time_bias_corrections->EETimeCorrAmplitudeBins.size(),
            hipMemcpyHostToDevice);
        hipMemcpy(d_data.EETimeCorrShiftBins, 
            h_data.time_bias_corrections->EETimeCorrShiftBins.data(),
            sizeof(float) * h_data.time_bias_corrections->EETimeCorrShiftBins.size(),
            hipMemcpyHostToDevice);
    }
    hipMemcpy(d_data.bxs, h_data.bxs,
        sizeof(BXVectorType),
        hipMemcpyHostToDevice);
    ecal::cuda::assert_if_error();

    int nthreads_per_block = conf.threads.x;
    int nblocks = (h_data.digis->size() + nthreads_per_block - 1) / nthreads_per_block;

    // 
    // 1d preparation kernel
    //
    unsigned int nchannels_per_block = 32;
    unsigned int threads_1d = 10 * nchannels_per_block;
    unsigned int blocks_1d = threads_1d > 10*h_data.digis->size() 
        ? 1 : (h_data.digis->size()*10 + threads_1d - 1) / threads_1d;
    int shared_bytes = nchannels_per_block * EcalDataFrame::MAXSAMPLES * (
        sizeof(bool) + sizeof(bool) + sizeof(bool) + sizeof(bool) + sizeof(char)
        + sizeof(bool)
    );
    std::cout << "nchannels = " << h_data.digis->size() << std::endl;
    std::cout << "shared memory per block = " << shared_bytes << "B\n";
    kernel_prep_1d_and_initialize<<<blocks_1d, threads_1d, shared_bytes>>>(
        d_data.pulses, d_data.epulses,
        d_data.digis_data, d_data.samples,
        d_data.amplitudes,
        d_data.gainsNoise,
        d_data.gainsPedestal,
        d_data.mean_x1,
        d_data.mean_x12,
        d_data.rms_x12,
        d_data.mean_x6,
        d_data.gain6Over1,
        d_data.gain12Over6,
        d_data.hasSwitchToGain6,
        d_data.hasSwitchToGain1,
        d_data.isSaturated,
        d_data.energies,
        d_data.chi2,
        d_data.pedestal,
        d_data.flags,
        d_data.acState,
        gainSwitchUseMaxSample,
        h_data.digis->size());
    ecal::cuda::assert_if_error();

    std::cout << " after kernel prep 1d\n";

    //
    // 2d preparation kernel
    //
    int blocks_2d = h_data.digis->size();
    dim3 threads_2d{10, 10};
    kernel_prep_2d<<<blocks_2d, threads_2d>>>(
        d_data.covariances, d_data.pulse_covariances,
        d_data.gainsNoise,
        d_data.rms_x12,
        d_data.rms_x6,
        d_data.rms_x1,
        d_data.gain12Over6,
        d_data.gain6Over1,
        d_data.G12SamplesCorrelation,
        d_data.G6SamplesCorrelation,
        d_data.G1SamplesCorrelation,
        d_data.noisecov,
        d_data.pulse_matrix,
        d_data.epulses,
        d_data.bxs,
        d_data.hasSwitchToGain6,
        d_data.hasSwitchToGain1,
        d_data.isSaturated);
    ecal::cuda::assert_if_error();

    std::cout << "after kernel prep 2d\n";

//#define ECAL_RECO_DEBUG_CPU4GPU
#ifdef ECAL_RECO_DEBUG_CPU4GPU

    // debug quantities before launching minimization
    std::vector<SampleVector> samples(h_data.digis->size());
    std::vector<PulseMatrixType> pulse_matrices(h_data.digis->size());
    std::vector<SampleMatrix> noisecovs(h_data.digis->size());
    std::vector<FullSampleMatrix> pulse_covariances(h_data.digis->size());
    std::vector<SampleVector> amplitudes(h_data.digis->size());
    std::vector<float> energies(h_data.digis->size());
    std::vector<char> statuses(h_data.digis->size());
    std::vector<float> chi2s(h_data.digis->size());
    std::vector<char> isSaturated(h_data.digis->size());
    std::vector<char> hasSwitchToGain6(h_data.digis->size());
    std::vector<char> hasSwitchToGain1(h_data.digis->size());
    hipMemcpy(samples.data(), d_data.samples,
        h_data.digis->size() * sizeof(SampleVector),
        hipMemcpyDeviceToHost);
    hipMemcpy(pulse_matrices.data(), d_data.pulse_matrix,
        pulse_matrices.size() * sizeof(PulseMatrixType),
        hipMemcpyDeviceToHost);
    hipMemcpy(noisecovs.data(), d_data.noisecov,
        noisecovs.size() * sizeof(SampleMatrix),
        hipMemcpyDeviceToHost);
    hipMemcpy(pulse_covariances.data(), d_data.pulse_covariances,
        pulse_covariances.size() * sizeof(FullSampleMatrix),
        hipMemcpyDeviceToHost);
    hipMemcpy(isSaturated.data(), 
        d_data.isSaturated,
        isSaturated.size() * sizeof(bool),
        hipMemcpyDeviceToHost);
    hipMemcpy(hasSwitchToGain6.data(), d_data.hasSwitchToGain6,
        hasSwitchToGain6.size() * sizeof(bool),
        hipMemcpyDeviceToHost);
    hipMemcpy(hasSwitchToGain1.data(), d_data.hasSwitchToGain1,
        hasSwitchToGain1.size() * sizeof(bool),
        hipMemcpyDeviceToHost);

    //std::cout << "dumping gpu quantities\n";

    cpu::kernel_minimize(
        noisecovs.data(),
        pulse_covariances.data(),
        h_data.bxs,
        samples.data(),
        amplitudes.data(),
        energies.data(),
        pulse_matrices.data(),
        reinterpret_cast<bool*>(statuses.data()),
        chi2s.data(),
        reinterpret_cast<bool*>(isSaturated.data()),
        reinterpret_cast<bool*>(hasSwitchToGain6.data()),
        reinterpret_cast<bool*>(hasSwitchToGain1.data()),
        h_data.digis->size(),
        50,
        gainSwitchUseMaxSample
    );

#endif

    hipEvent_t start_event;
    hipEvent_t end_event;
    hipEventCreate(&start_event);
    hipEventCreate(&end_event);

    hipEventRecord(start_event, 0);
    if (conf.runV1)
        v1::minimization_procedure(d_data, h_data, conf);
    else
        v2::minimization_procedure(d_data, h_data);
    hipEventRecord(end_event, 0);
    hipEventSynchronize(end_event);
    float ms;
    hipEventElapsedTime(&ms, start_event, end_event);
    std::cout << "elapsed time = " << ms << std::endl;
    ecal::cuda::assert_if_error();

    //
    // TODO: this guy can run concurrently with other kernels,
    // there is no dependence on the order of execution
    //
    unsigned int threads_time_init = threads_1d;
    unsigned int blocks_time_init = blocks_1d;
    int sharedBytesInit = 2 * threads_time_init * sizeof(SampleVector::Scalar);
    kernel_time_computation_init<<<blocks_time_init, threads_time_init,
                                   sharedBytesInit>>>(
        d_data.digis_data, d_data.ids,
        d_data.rms_x12,
        d_data.rms_x6,
        d_data.rms_x1,
        d_data.mean_x12,
        d_data.mean_x6,
        d_data.mean_x1,
        d_data.gain12Over6,
        d_data.gain6Over1,
        d_data.sample_values,
        d_data.sample_value_errors,
        d_data.ampMaxError,
        d_data.useless_sample_values,
        d_data.pedestal_nums,
        barrel 
            ? h_data.sample_mask.getEcalSampleMaskRecordEB()
            : h_data.sample_mask.getEcalSampleMaskRecordEE(),
        h_data.digis->size()
    );
    ecal::cuda::assert_if_error();

    // 
    // TODO: small kernel only for EB. It needs to be checked if 
    /// fusing such small kernels is beneficial in here
    //
    if (barrel) {
        kernel_time_compute_fixMGPAslew<<<blocks_time_init, threads_time_init>>>(
            d_data.digis_data,
            d_data.sample_values,
            d_data.sample_value_errors,
            d_data.useless_sample_values,
            h_data.sample_mask.getEcalSampleMaskRecordEB(),
            h_data.digis->size()
        );
        ecal::cuda::assert_if_error();
    }

    //
    // 
    //
    int sharedBytes = EcalDataFrame::MAXSAMPLES * nchannels_per_block *
        4 * sizeof(SampleVector::Scalar);
    auto const threads_nullhypot = threads_1d;
    auto const blocks_nullhypot = blocks_1d;
    kernel_time_compute_nullhypot<<<blocks_nullhypot, threads_nullhypot, 
                                    sharedBytes>>>(
        d_data.sample_values,
        d_data.sample_value_errors,
        d_data.useless_sample_values,
        d_data.chi2sNullHypot,
        d_data.sum0sNullHypot,
        d_data.sumAAsNullHypot,
        h_data.digis->size()
    );
    ecal::cuda::assert_if_error();

    std::cout << "*** before makeratio kernel ***\n";
    //
    // TODO: configurable parameters for launch context below
    //
    unsigned int nchannels_per_block_makeratio = 10;
    unsigned int threads_makeratio = 45 * nchannels_per_block_makeratio;
    unsigned int blocks_makeratio = threads_makeratio > 45 * h_data.digis->size()
        ? 1
        : (h_data.digis->size() * 45 + threads_makeratio - 1) / threads_makeratio;
    int sharedBytesMakeRatio = 5 * threads_makeratio * sizeof(SampleVector::Scalar);
    kernel_time_compute_makeratio<<<blocks_makeratio, threads_makeratio,
                                    sharedBytesMakeRatio>>>(
        d_data.sample_values,
        d_data.sample_value_errors,
        d_data.useless_sample_values,
        d_data.pedestal_nums,
        barrel ? d_data.amplitudeFitParametersEB : d_data.amplitudeFitParametersEE,
        barrel ? d_data.timeFitParametersEB : d_data.timeFitParametersEE,
        d_data.sumAAsNullHypot,
        d_data.sum0sNullHypot,
        d_data.tMaxAlphaBetas,
        d_data.tMaxErrorAlphaBetas,
        d_data.accTimeMax,
        d_data.accTimeWgt,
        d_data.tcState,
        barrel ? d_data.timeFitParametersSizeEB : d_data.timeFitParametersSizeEE,
        barrel ? d_data.timeFitLimitsFirstEB : d_data.timeFitLimitsFirstEE,
        barrel ? d_data.timeFitLimitsSecondEB : d_data.timeFitLimitsSecondEE,
        h_data.digis->size()
    );
    ecal::cuda::assert_if_error();

    //
    //
    //
    auto const threads_findamplchi2 = threads_1d;
    auto const blocks_findamplchi2 = blocks_1d;
    int const sharedBytesFindAmplChi2 = 2 * threads_findamplchi2 * 
        sizeof(SampleVector::Scalar);
    kernel_time_compute_findamplchi2_and_finish<<<blocks_findamplchi2,
                                       threads_findamplchi2,
                                       sharedBytesFindAmplChi2>>>(
        d_data.sample_values,
        d_data.sample_value_errors,
        d_data.useless_sample_values,
        d_data.tMaxAlphaBetas,
        d_data.tMaxErrorAlphaBetas,
        d_data.accTimeMax,
        d_data.accTimeWgt,
        barrel ? d_data.amplitudeFitParametersEB : d_data.amplitudeFitParametersEE,
        d_data.sumAAsNullHypot,
        d_data.sum0sNullHypot,
        d_data.chi2sNullHypot,
        d_data.tcState,
        d_data.ampMaxAlphaBeta,
        d_data.ampMaxError,
        d_data.timeMax,
        d_data.timeError,
        h_data.digis->size()
    );
    ecal::cuda::assert_if_error();

    //
    //
    //
    auto const threads_ampl = threads_1d;
    auto const blocks_ampl = blocks_1d;
    int const sharedBytesAmpl = 5 * threads_ampl * sizeof(SampleVector::Scalar);
    kernel_time_compute_ampl<<<blocks_ampl, threads_ampl,
                               sharedBytesAmpl>>>(
        d_data.sample_values,
        d_data.sample_value_errors,
        d_data.useless_sample_values,
        d_data.timeMax,
        barrel ? d_data.amplitudeFitParametersEB : d_data.amplitudeFitParametersEE,
        d_data.amplitudeMax,
        h_data.digis->size()
    );
    ecal::cuda::assert_if_error();

    //
    //
    //
    auto const threads_timecorr = 32;
    auto const blocks_timecorr = threads_timecorr > h_data.digis->size()
        ? 1 : (h_data.digis->size() + threads_timecorr-1) / threads_timecorr;
    kernel_time_correction_and_finalize<<<blocks_timecorr, threads_timecorr>>>(
        d_data.energies,
        d_data.digis_data,
        barrel ? d_data.EBTimeCorrAmplitudeBins : d_data.EETimeCorrAmplitudeBins,
        barrel ? d_data.EBTimeCorrShiftBins : d_data.EETimeCorrShiftBins,
        d_data.timeMax,
        d_data.timeError,
        d_data.rms_x12,
        d_data.timeCalibConstants,
        d_data.jitter,
        d_data.jitterError,
        d_data.flags,
        barrel 
            ? h_data.time_bias_corrections->EBTimeCorrAmplitudeBins.size() 
            : h_data.time_bias_corrections->EETimeCorrAmplitudeBins.size(),
        barrel 
            ? d_data.timeConstantTermEB
            : d_data.timeConstantTermEE,
        d_data.offsetTimeValue,
        barrel 
            ? d_data.timeNconstEB
            : d_data.timeNconstEE,
        barrel 
            ? d_data.amplitudeThreshEB
            : d_data.amplitudeThreshEE,
        barrel
            ? d_data.outOfTimeThreshG12pEB
            : d_data.outOfTimeThreshG12pEE,
        barrel 
            ? d_data.outOfTimeThreshG12mEB
            : d_data.outOfTimeThreshG12mEE,
        barrel
            ? d_data.outOfTimeThreshG61pEB
            : d_data.outOfTimeThreshG61pEE,
        barrel
            ? d_data.outOfTimeThreshG61mEB
            : d_data.outOfTimeThreshG61mEE,
        h_data.digis->size()
    );
    ecal::cuda::assert_if_error();

    //
    // transfer the results back
    //
//    h_data.rechits_soa.amplitude = std::move(energies);
//    h_data.rechits_soa.chi2 = std::move(chi2s);
    hipMemcpy(&(*h_data.rechits_soa.amplitude.begin()),
               d_data.energies,
               h_data.rechits_soa.amplitude.size() * sizeof(float),
               hipMemcpyDeviceToHost);
    hipMemcpy(h_data.rechits_soa.pedestal.data(),
               d_data.pedestal,
               h_data.rechits_soa.pedestal.size() * sizeof(float),
               hipMemcpyDeviceToHost);
    hipMemcpy(&(*h_data.rechits_soa.chi2.begin()),
               d_data.chi2,
               h_data.rechits_soa.chi2.size() * sizeof(float),
               hipMemcpyDeviceToHost);
    hipMemcpy(&(*h_data.rechits_soa.did.begin()),
               d_data.ids,
               h_data.rechits_soa.did.size() * sizeof(uint32_t),
               hipMemcpyDeviceToHost);
    hipMemcpy(h_data.rechits_soa.flags.data(),
               d_data.flags,
               h_data.rechits_soa.flags.size() * sizeof(uint32_t),
               hipMemcpyDeviceToHost);
    hipMemcpy(h_data.rechits_soa.jitter.data(),
               d_data.jitter,
               h_data.rechits_soa.jitter.size() * sizeof(float),
               hipMemcpyDeviceToHost);
    hipMemcpy(h_data.rechits_soa.jitterError.data(),
               d_data.jitterError,
               h_data.rechits_soa.jitterError.size() * sizeof(float),
               hipMemcpyDeviceToHost);
    hipMemcpy(h_data.rechits_soa.amplitudesAll.data(),
               d_data.amplitudes,
               h_data.rechits_soa.amplitudesAll.size() * 
               sizeof(::ecal::reco::ComputationScalarType),
               hipMemcpyDeviceToHost);
}

}}
