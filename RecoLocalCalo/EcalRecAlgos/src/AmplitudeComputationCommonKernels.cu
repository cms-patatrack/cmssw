#include "hip/hip_runtime.h"
#include <iostream>
#include <limits>

#include "hip/hip_runtime.h"

#include "DataFormats/EcalDigi/interface/EcalDataFrame.h"
#include "DataFormats/Math/interface/approx_exp.h"
#include "DataFormats/Math/interface/approx_log.h"

#include "CondFormats/EcalObjects/interface/EcalPulseShapes.h"
#include "CondFormats/EcalObjects/interface/EcalPulseCovariances.h"

#include "inplace_fnnls.h"
#include "AmplitudeComputationKernelsV1.h"

namespace ecal { namespace multifit {

///
/// assume kernel launch configuration is 
/// (MAXSAMPLES * nchannels, blocks)
/// TODO: is there a point to split this kernel further to separate reductions
/// 
__global__
void kernel_prep_1d_and_initialize(EcalPulseShape const* shapes_in,
                    FullSampleVector* shapes_out, 
                    uint16_t const* digis_in,
                    SampleVector* amplitudes,
                    SampleVector* amplitudesForMinimization,
                    SampleGainVector* gainsNoise,
                    SampleGainVector* gainsPedestal,
                    float const* mean_x1,
                    float const* mean_x12,
                    float const* rms_x12,
                    float const* mean_x6,
                    float const* gain6Over1,
                    float const* gain12Over6,
                    bool* hasSwitchToGain6,
                    bool* hasSwitchToGain1,
                    bool* isSaturated,
                    float* energies,
                    float* chi2,
                    char* acState,
                    bool gainSwitchUseMaxSample,
                    int nchannels) {
    constexpr bool dynamicPedestal = false;
    constexpr int nsamples = EcalDataFrame::MAXSAMPLES;
    constexpr int sample_max = 5;
    constexpr int full_pulse_max = 9;
    int tx = threadIdx.x + blockIdx.x*blockDim.x;
    int nchannels_per_block = blockDim.x / nsamples;
    int total_threads = nchannels * nsamples;
    int ch = tx / nsamples;

    if (ch < nchannels) {
        // array of 10 x channels per block
        // TODO: any other way of doing simple reduction
        // assume bool is 1 byte, should be quite safe
        extern __shared__ char shared_mem[];
        bool* shr_hasSwitchToGain6 = reinterpret_cast<bool*>(
            shared_mem);
        bool* shr_hasSwitchToGain1 = shr_hasSwitchToGain6 + 
            nchannels_per_block*nsamples;
        bool* shr_hasSwitchToGain0 = shr_hasSwitchToGain1 + 
            nchannels_per_block*nsamples;
        bool* shr_isSaturated = shr_hasSwitchToGain0 + 
            nchannels_per_block*nsamples;
        bool* shr_hasSwitchToGain0_tmp = shr_isSaturated + 
            nchannels_per_block*nsamples;
        char* shr_counts = reinterpret_cast<char*>(
            shr_hasSwitchToGain0_tmp) + nchannels_per_block*nsamples;

        //
        // pulse shape template
        //
        int sample = threadIdx.x % nsamples;
        for (int isample=sample; isample<EcalPulseShape::TEMPLATESAMPLES; 
            isample+=nsamples)
            shapes_out[ch](isample + 7) = shapes_in[ch].pdfval[isample];

        //
        // amplitudes
        //
        int adc = ecal::mgpa::adc(digis_in[tx]);
        int gainId = ecal::mgpa::gainId(digis_in[tx]);
        auto const rmsForChecking = rms_x12[ch];
        SampleVector::Scalar amplitude = 0.;
        SampleVector::Scalar pedestal = 0.;
        SampleVector::Scalar gainratio = 0.;

        shr_hasSwitchToGain6[threadIdx.x] = gainId == EcalMgpaBitwiseGain6;
        shr_hasSwitchToGain1[threadIdx.x] = gainId == EcalMgpaBitwiseGain1;
        shr_hasSwitchToGain0_tmp[threadIdx.x] = gainId == EcalMgpaBitwiseGain0;
        shr_hasSwitchToGain0[threadIdx.x] = shr_hasSwitchToGain0_tmp[threadIdx.x];
        shr_counts[threadIdx.x] = 0;
        __syncthreads();
        
        // non-divergent branch (except for the last 4 threads)
        if (threadIdx.x<=blockDim.x-5) {
            #pragma unroll
            for (int i=0; i<5; i++)
                shr_counts[threadIdx.x] += 
                    shr_hasSwitchToGain0[threadIdx.x+i];
        }
        shr_isSaturated[threadIdx.x] = shr_counts[threadIdx.x] == 5;

        //
        // unrolled reductions
        // TODO
        //
        if (sample < 5) {
            shr_hasSwitchToGain6[threadIdx.x] = 
                shr_hasSwitchToGain6[threadIdx.x] ||
                shr_hasSwitchToGain6[threadIdx.x + 5];
            shr_hasSwitchToGain1[threadIdx.x] =
                shr_hasSwitchToGain1[threadIdx.x] ||
                shr_hasSwitchToGain1[threadIdx.x + 5];
            
            // duplication of hasSwitchToGain0 in order not to
            // introduce another syncthreads
            shr_hasSwitchToGain0_tmp[threadIdx.x] = 
                shr_hasSwitchToGain0_tmp[threadIdx.x] || 
                shr_hasSwitchToGain0_tmp[threadIdx.x+5];
        }
        __syncthreads();
        
        if (sample<2) {
            // note, both threads per channel take value [3] twice to avoid another if
            shr_hasSwitchToGain6[threadIdx.x] = 
                shr_hasSwitchToGain6[threadIdx.x] ||
                shr_hasSwitchToGain6[threadIdx.x+2] || 
                shr_hasSwitchToGain6[threadIdx.x+3];
            shr_hasSwitchToGain1[threadIdx.x] =
                shr_hasSwitchToGain1[threadIdx.x] ||
                shr_hasSwitchToGain1[threadIdx.x+2] || 
                shr_hasSwitchToGain1[threadIdx.x+3];

            shr_hasSwitchToGain0_tmp[threadIdx.x] = 
                shr_hasSwitchToGain0_tmp[threadIdx.x] ||
                shr_hasSwitchToGain0_tmp[threadIdx.x+2] || 
                shr_hasSwitchToGain0_tmp[threadIdx.x+3];

            // sample < 2 -> first 2 threads of each channel will be used here
            // => 0 -> will compare 3 and 4
            // => 1 -> will compare 4 and 5
            shr_isSaturated[threadIdx.x+3] = 
                shr_isSaturated[threadIdx.x+3] || shr_isSaturated[threadIdx.x+4];
        }
        __syncthreads();

        bool check_hasSwitchToGain0 = false;

        if (sample==0) {
            shr_hasSwitchToGain6[threadIdx.x] = 
                shr_hasSwitchToGain6[threadIdx.x] || 
                shr_hasSwitchToGain6[threadIdx.x+1];
            shr_hasSwitchToGain1[threadIdx.x] = 
                shr_hasSwitchToGain1[threadIdx.x] ||
                shr_hasSwitchToGain1[threadIdx.x+1];
            shr_hasSwitchToGain0_tmp[threadIdx.x] =
                shr_hasSwitchToGain0_tmp[threadIdx.x] ||
                shr_hasSwitchToGain0_tmp[threadIdx.x+1];

            hasSwitchToGain6[ch] = shr_hasSwitchToGain6[threadIdx.x];
            hasSwitchToGain1[ch] = shr_hasSwitchToGain1[threadIdx.x];

            // set only for the threadIdx.x corresponding to sample==0
            check_hasSwitchToGain0 = shr_hasSwitchToGain0_tmp[threadIdx.x];

            shr_isSaturated[threadIdx.x+3] = 
                shr_isSaturated[threadIdx.x+3] || 
                shr_isSaturated[threadIdx.x+4];
            isSaturated[ch] = shr_isSaturated[threadIdx.x+3];
        }

        // TODO: divergent branch
        if (gainId==0 || gainId==3) {
            pedestal = mean_x1[ch];
            gainratio = gain6Over1[ch] * gain12Over6[ch];
            gainsNoise[ch](sample) = 2;
            gainsPedestal[ch](sample) = dynamicPedestal ? 2 : -1;
        } else if (gainId==1) {
            pedestal = mean_x12[ch];
            gainratio = 1.;
            gainsNoise[ch](sample) = 0;
            gainsPedestal[ch](sample) = dynamicPedestal ? 0 : -1;
        } else if (gainId==2) {
            pedestal = mean_x6[ch];
            gainratio = gain12Over6[ch];
            gainsNoise[ch](sample)  = 1;
            gainsPedestal[ch](sample) = dynamicPedestal ? 1 : -1;
        }

        // TODO: compile time constant -> branch should be non-divergent
        if (dynamicPedestal)
            amplitude = static_cast<SampleVector::Scalar>(adc) * gainratio;
        else
            amplitude = (static_cast<SampleVector::Scalar>(adc) - pedestal) * gainratio;

        amplitudes[ch][sample] = amplitude;

#ifdef ECAL_RECO_CUDA_DEBUG
        printf("%d %d %d %d %f %f %f\n", tx, ch, sample, adc, amplitude,
            pedestal, gainratio);
        if (adc==0)
            printf("adc is zero\n");
#endif

        //
        // initialization
        //
        amplitudesForMinimization[ch](sample) = 0;

        if (sample == 0) {
            //
            // initialization
            //
            acState[ch] = static_cast<char>(MinimizationState::NotFinished);
            energies[ch] = 0;
            chi2[ch] = 0;

            // this corresponds to cpu branching on lastSampleBeforeSaturation
            // likely false
            if (check_hasSwitchToGain0) {
                // assign for the case some sample having gainId == 0
                energies[ch] = amplitudes[ch][sample_max];

                // check if samples before sample_max have true
                bool saturated_before_max = false;
                #pragma unroll
                for (char ii=0; ii<5; ii++)
                    saturated_before_max = saturated_before_max ||
                        shr_hasSwitchToGain0[threadIdx.x + ii];

                // if saturation is in the max sample and not in the first 5
                if (!saturated_before_max && 
                    shr_hasSwitchToGain0[threadIdx.x + sample_max])
                    energies[ch] = 49140; // 4095 * 12

                // set state flag to terminate further processing of this channel
                acState[ch] = static_cast<char>(MinimizationState::Precomputed); 
                return;
            }

            // according to cpu version
            auto max_amplitude = amplitudes[ch][sample_max]; 
            // according to cpu version
            auto shape_value = shapes_out[ch](full_pulse_max); 
            // note, no syncing as the same thread will be accessing here
            bool hasGainSwitch = shr_hasSwitchToGain6[threadIdx.x]
                || shr_hasSwitchToGain1[threadIdx.x]
                || shr_isSaturated[threadIdx.x+3];
            if (hasGainSwitch && gainSwitchUseMaxSample) {
                // thread for sample=0 will access the right guys
                energies[ch] = max_amplitude / shape_value;
                acState[ch] = static_cast<char>(MinimizationState::Precomputed);
            }
            
            // this happens cause sometimes rms_x12 is 0...
            // needs to be checkec why this is the case
            // general case here is that noisecov is a Zero matrix
            if (rmsForChecking == 0) {
                acState[ch] = static_cast<char>(MinimizationState::Precomputed);
                return;
            }
        }
    }
}

///
/// assume kernel launch configuration is 
/// ([MAXSAMPLES, MAXSAMPLES], nchannels)
///
__global__
void kernel_prep_2d(EcalPulseCovariance const* pulse_cov_in,
                    FullSampleMatrix* pulse_cov_out,
                    SampleGainVector const* gainNoise,
                    SampleMatrixD const* noisecorrs,
                    float const* rms_x12,
                    float const* rms_x6,
                    float const* rms_x1,
                    float const* gain12Over6,
                    float const* gain6Over1,
                    SampleMatrix* noisecov,
                    PulseMatrixType* pulse_matrix,
                    FullSampleVector const* pulse_shape,
                    BXVectorType const* bxs,
                    bool const* hasSwitchToGain6,
                    bool const* hasSwitchToGain1,
                    bool const* isSaturated) {
    int ch = blockIdx.x;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    constexpr int nsamples = EcalDataFrame::MAXSAMPLES;
    constexpr float addPedestalUncertainty = 0.f;
    constexpr bool dynamicPedestal = false;
    constexpr bool simplifiedNoiseModelForGainSwitch = true;
    constexpr int template_samples = EcalPulseShape::TEMPLATESAMPLES;

    // only ty == 0 and 1 will go for a second iteration
    for (int iy=ty; iy<template_samples; iy+=nsamples)
        for (int ix=tx; ix<template_samples; ix+=nsamples)
            pulse_cov_out[ch](iy+7, ix+7) = pulse_cov_in[ch].covval[iy][ix];

    /*
    for (int iy=ty, ix=tx; ix<=template_samples && iy<=template_samples; 
        ix+=nsamples, iy+=nsamples)
        pulse_cov_out[ch](iy+7, ix+7) = pulse_cov_in[ch].covval[iy][ix];
        */
    
    bool tmp0 = hasSwitchToGain6[ch];
    bool tmp1 = hasSwitchToGain1[ch];
    bool tmp2 = isSaturated[ch];
    bool hasGainSwitch = tmp0 || tmp1 || tmp2;
    // non-divergent branch for all threads per block
    if (hasGainSwitch) {
        // TODO: did not include simplified noise model
        float noise_value = 0;

        // non-divergent branch - all threads per block
        // TODO: all of these constants indicate that 
        // that these parts could be splitted into completely different 
        // kernels and run one of them only depending on the config
        if (simplifiedNoiseModelForGainSwitch) {
            int isample_max = 5; // according to cpu defs
            int gainidx = gainNoise[ch][isample_max];

            // non-divergent branches
            if (gainidx==0)
                noise_value = rms_x12[ch]*rms_x12[ch]*noisecorrs[0](ty, tx);
            if (gainidx==1) 
                noise_value = gain12Over6[ch]*gain12Over6[ch] * rms_x6[ch]*rms_x6[ch]
                    *noisecorrs[1](ty, tx);
            if (gainidx==2)
                noise_value = gain12Over6[ch]*gain12Over6[ch]
                    * gain6Over1[ch]*gain6Over1[ch] * rms_x1[ch]*rms_x1[ch]
                    * noisecorrs[2](ty, tx);
            if (!dynamicPedestal && addPedestalUncertainty>0.f)
                noise_value += addPedestalUncertainty*addPedestalUncertainty;
        } else {
            // 
            int gainidx=0;
            char mask = gainidx;
            int pedestal = gainNoise[ch][ty] == mask ? 1 : 0;
            noise_value += /* gainratio is 1*/ rms_x12[ch]*rms_x12[ch]
                *pedestal*noisecorrs[0](ty, tx);
            // non-divergent branch
            if (!dynamicPedestal && addPedestalUncertainty>0.f) {
                noise_value += /* gainratio is 1 */
                    addPedestalUncertainty*addPedestalUncertainty*pedestal;
            }

            //
            gainidx=1;
            mask = gainidx;
            pedestal = gainNoise[ch][ty] == mask ? 1 : 0;
            noise_value += gain12Over6[ch]*gain12Over6[ch]
                *rms_x6[ch]*rms_x6[ch]*pedestal*noisecorrs[1](ty, tx);
            // non-divergent branch
            if (!dynamicPedestal && addPedestalUncertainty>0.f) {
                noise_value += gain12Over6[ch]*gain12Over6[ch]
                    *addPedestalUncertainty*addPedestalUncertainty
                    *pedestal;
            }
            
            //
            gainidx=2;
            mask = gainidx;
            pedestal = gainNoise[ch][ty] == mask ? 1 : 0;
            float tmp = gain6Over1[ch] * gain12Over6[ch];
            noise_value += tmp*tmp * rms_x1[ch]*rms_x1[ch]
                *pedestal*noisecorrs[2](ty, tx);
            // non-divergent branch
            if (!dynamicPedestal && addPedestalUncertainty>0.f) {
                noise_value += tmp*tmp * addPedestalUncertainty*addPedestalUncertainty
                    * pedestal;
            }
        }

        noisecov[ch](ty, tx) = noise_value;
    } else {
        auto rms = rms_x12[ch];
        float noise_value = rms*rms * noisecorrs[0](ty, tx);
        if (!dynamicPedestal && addPedestalUncertainty>0.f)
            noise_value += addPedestalUncertainty*addPedestalUncertainty;
        noisecov[ch](ty, tx) = noise_value;
    }

    // pulse matrix
    int bx = (*bxs)(tx);
    int offset = 7 - 3 - bx;
    float value = pulse_shape[ch](offset + ty);
    pulse_matrix[ch](ty, tx) = value;
}

__global__
void kernel_permute_results(
        SampleVector *amplitudes,
        BXVectorType const*activeBXs,
        float *energies,
        char const* acState,
        int const nchannels) {
    // constants
    constexpr int nsamples = EcalDataFrame::MAXSAMPLES;

    // indices
    int const tx = threadIdx.x + blockIdx.x * blockDim.x;
    int const ch = tx / nsamples;
    int const iii = tx % nsamples; // this is to address activeBXs

    if (ch >= nchannels) return;

    // configure shared memory and cp into it
    extern __shared__ char smem[];
    SampleVector::Scalar* values = reinterpret_cast<SampleVector::Scalar*>(
        smem);
    values[threadIdx.x] = amplitudes[ch](iii);
    __syncthreads();

    // get the sample for this bx
    auto const sample = static_cast<int>(activeBXs[ch](iii)) + 5;
    auto const state = static_cast<MinimizationState>(acState[ch]);

    // store back to global
    amplitudes[ch](sample) = values[threadIdx.x];

    // store sample 5 separately
    // only for the case when minimization was performed
    // not for cases with precomputed amplitudes
    if (sample == 5 && state != MinimizationState::Precomputed)
        energies[ch] = values[threadIdx.x];
}

///
/// Build an Ecal RecHit.
/// TODO: Use SoA data structures on the host directly
/// the reason for removing this from minimize kernel is to isolate the minimize + 
/// again, building an aos rec hit involves strides... -> bad memory access pattern
///
#ifdef RUN_BUILD_AOS_RECHIT
__global__
void kernel_build_rechit(
    float const* energies,
    float const* chi2s,
    uint32_t* dids,
    EcalUncalibratedRecHit* rechits,
    int nchannels) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < nchannels) {
        rechits[idx] = EcalUncalibratedRecHit{dids[idx], energies[idx],
            0, 0, chi2s[idx], 0};
    }
}
#endif

}}
