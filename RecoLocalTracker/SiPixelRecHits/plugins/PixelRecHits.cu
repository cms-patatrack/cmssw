#include "hip/hip_runtime.h"
// C++ headers
#include <algorithm>
#include <numeric>

// CUDA runtime
#include <hip/hip_runtime.h>

// CMSSW headers
#include "FWCore/MessageLogger/interface/MessageLogger.h"
#include "RecoLocalTracker/SiPixelClusterizer/plugins/SiPixelRawToClusterGPUKernel.h"
#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "RecoLocalTracker/SiPixelClusterizer/plugins/gpuClusteringConstants.h"
#include "PixelRecHits.h"
#include "gpuPixelRecHits.h"

namespace {
  __global__
  void setHitsLayerStart(uint32_t const * __restrict__ hitsModuleStart, pixelCPEforGPU::ParamsOnGPU const * cpeParams, uint32_t* hitsLayerStart) {
    auto i = blockIdx.x * blockDim.x + threadIdx.x;

    assert(0==hitsModuleStart[0]);

    if(i < 11) {
      hitsLayerStart[i] = hitsModuleStart[cpeParams->layerGeometry().layerStart[i]];
#ifdef GPU_DEBUG
      printf ("LayerStart %d %d: %d\n",i, cpeParams->layerGeometry().layerStart[i], hitsLayerStart[i]);
#endif
    }
  }
}

namespace pixelgpudetails {

  void PixelRecHitGPUKernel::makeHitsAsync(
                                           TrackingRecHit2DCUDA & hits_d,
                                           SiPixelDigisCUDA const& digis_d,
                                           SiPixelClustersCUDA const& clusters_d,
                                           BeamSpotCUDA const& bs_d,
                                           pixelCPEforGPU::ParamsOnGPU const * cpeParams,
                                           cuda::stream_t<>& stream
                                          ) const {


    int threadsPerBlock = 256;
    int blocks = digis_d.nModules(); // active modules (with digis)

#ifdef GPU_DEBUG
    std::cout << "launching getHits kernel for " << blocks << " blocks" << std::endl;
#endif
    if(blocks)  // protect from empty events
    gpuPixelRecHits::getHits<<<blocks, threadsPerBlock, 0, stream.id()>>>(
      cpeParams,
      bs_d.data(),
      digis_d.moduleInd(),
      digis_d.xx(), digis_d.yy(), digis_d.adc(),
      clusters_d.moduleStart(),
      clusters_d.clusInModule(), clusters_d.moduleId(),
      digis_d.clus(),
      digis_d.nDigis(),
      clusters_d.clusModuleStart(),
      hits_d.view()
    );
    cudaCheck(hipGetLastError());

     
    // assuming full warp of threads is better than a smaller number...
    setHitsLayerStart<<<1, 32, 0, stream.id()>>>(clusters_d.clusModuleStart(), cpeParams, hits_d.hitsLayerStart());
    cudaCheck(hipGetLastError());

    auto nhits_ = clusters_d.nClusters();
    if (nhits_ >= TrackingRecHit2DSOAView::maxHits()) {
      edm::LogWarning("PixelRecHitGPUKernel" ) << "Hits Overflow " << nhits_  << " > " << TrackingRecHit2DSOAView::maxHits();
    } 

    if (nhits_)
    cudautils::fillManyFromVector(hits_d.phiBinner(), hits_d.phiBinnerWS(), 10, hits_d.iphi(), hits_d.hitsLayerStart(), nhits_, 256, stream.id());
    cudaCheck(hipGetLastError());
  }
}
