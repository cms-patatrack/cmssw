#include "PixelRecHits.h"
#include "gpuPixelRecHits.h"

#include "RecoLocalTracker/SiPixelClusterizer/plugins/gpuClustering.h"

#include "EventFilter/SiPixelRawToDigi/plugins/RawToDigiGPU.h" // for context....
#include "EventFilter/SiPixelRawToDigi/plugins/cudaCheck.h"

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <numeric>
#include <algorithm>

HitsOnGPU allocHitsOnGPU() {
   HitsOnGPU hh;
   cudaCheck(hipMalloc((void**) & hh.hitsModuleStart_d,(gpuClustering::MaxNumModules+1)*sizeof(uint32_t)));
   cudaCheck(hipMalloc((void**) & hh.xg_d,(gpuClustering::MaxNumModules*256)*sizeof(float)));
   cudaCheck(hipMalloc((void**) & hh.yg_d,(gpuClustering::MaxNumModules*256)*sizeof(float)));
   cudaCheck(hipMalloc((void**) & hh.zg_d,(gpuClustering::MaxNumModules*256)*sizeof(float)));
   hipDeviceSynchronize();

   return hh;
}


void pixelRecHits_wrapper(
      context const & c,
      pixelCPEforGPU::ParamsOnGPU const * cpeParams,
      uint32_t ndigis,
      uint32_t nModules, // active modules (with digis)
      HitsOnGPU & hh
)
{

 
 uint32_t hitsModuleStart[gpuClustering::MaxNumModules+1];
 hitsModuleStart[0] =0;
 cudaCheck(hipMemcpyAsync(&hitsModuleStart[1], c.clusInModule_d, gpuClustering::MaxNumModules*sizeof(uint32_t), hipMemcpyDeviceToHost, c.stream)); 

 std::partial_sum(std::begin(hitsModuleStart),std::end(hitsModuleStart),std::begin(hitsModuleStart));

 auto nhits = hitsModuleStart[gpuClustering::MaxNumModules];
 std::cout << " total number of clusters " << nhits << std::endl;

 cudaCheck(hipMemcpyAsync(hh.hitsModuleStart_d, &hitsModuleStart, (gpuClustering::MaxNumModules+1)*sizeof(uint32_t), hipMemcpyHostToDevice, c.stream));

  
 int threadsPerBlock = 256;
 int blocks = nModules;
 gpuPixelRecHits::getHits<<<blocks, threadsPerBlock, 0, c.stream>>>(
               cpeParams,
               c.moduleInd_d,
               c.xx_d, c.yy_d, c.adc_d,
               c.moduleStart_d,
               c.clusInModule_d, c.moduleId_d,
               c.clus_d,
               ndigis,
               hh.hitsModuleStart_d,
               hh.xg_d,hh.yg_d,hh.zg_d,
               false
  );

}
