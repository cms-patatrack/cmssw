#include "gpuClusterTracks.h"

namespace gpuVertexFinder {


  void Producer::allocateOnGPU() {
    cudaCheck(hipMalloc(&onGPU.zt, OnGPU::MAXTRACKS*sizeof(float)));
    cudaCheck(hipMalloc(&onGPU.ezt2, OnGPU::MAXTRACKS*sizeof(float)));
    cudaCheck(hipMalloc(&onGPU.iv, OnGPU::MAXTRACKS*sizeof(int32_t)));

    cudaCheck(hipMalloc(&onGPU.nv, sizeof(uint32_t)));
    cudaCheck(hipMalloc(&onGPU.zv, OnGPU::MAXVTX*sizeof(float)));
    cudaCheck(hipMalloc(&onGPU.wv, OnGPU::MAXVTX*sizeof(float)));
    cudaCheck(hipMalloc(&onGPU.chi2, OnGPU::MAXVTX*sizeof(float)));
 

    cudaCheck(hipMalloc(&onGPU.izt, OnGPU::MAXTRACKS*sizeof(int8_t)));
    cudaCheck(hipMalloc(&onGPU.nn, OnGPU::MAXTRACKS*sizeof(int32_t)));

    cudaCheck(hipMalloc(&onGPU_d,sizeof(OnGPU)));
    cudaCheck(hipMemcpy(onGPU_d,&onGPU,sizeof(OnGPU),hipMemcpyHostToDevice));

  }
	      
  void Producer::deallocateOnGPU() {
    cudaCheck(hipFree(onGPU.zt));
    cudaCheck(hipFree(onGPU.ezt2));
    cudaCheck(hipFree(onGPU.iv));

    cudaCheck(hipFree(onGPU.nv));
    cudaCheck(hipFree(onGPU.zv));
    cudaCheck(hipFree(onGPU.wv));
    cudaCheck(hipFree(onGPU.chi2));
 

    cudaCheck(hipFree(onGPU.izt));
    cudaCheck(hipFree(onGPU.nn));

    cudaCheck(hipFree(onGPU_d));

  }


  void Producer::produce(hipStream_t stream,
			 float const * zt,
			 float const * ezt2,
			 uint32_t ntrks
			 ) {
    

    cudaCheck(hipMemcpyAsync(onGPU.zt,zt,ntrks*sizeof(float),
			      hipMemcpyHostToDevice,stream));
    cudaCheck(hipMemcpyAsync(onGPU.ezt2,ezt2,ntrks*sizeof(float),
			      hipMemcpyHostToDevice,stream));
    
    assert(onGPU_d);
    clusterTracks<<<1,1024,0,stream>>>(ntrks,onGPU_d,minT,eps,errmax,chi2max);
    
    cudaCheck(hipMemcpyAsync(&gpuProduct.nVertices, onGPU.nv, sizeof(uint32_t),
			      hipMemcpyDeviceToHost, stream));
    
    gpuProduct.ivtx.resize(ntrks);
    cudaCheck(hipMemcpyAsync(gpuProduct.ivtx.data(),onGPU.iv,sizeof(int32_t)*ntrks,
			      hipMemcpyDeviceToHost, stream));
 

  }
  
  Producer::GPUProduct const & Producer::fillResults(hipStream_t stream) {

    // finish copy
    gpuProduct.z.resize(gpuProduct.nVertices);
    cudaCheck(hipMemcpyAsync(gpuProduct.z.data(),onGPU.zv,sizeof(float)*gpuProduct.nVertices,
			      hipMemcpyDeviceToHost, stream));
    gpuProduct.zerr.resize(gpuProduct.nVertices);
    cudaCheck(hipMemcpyAsync(gpuProduct.zerr.data(),onGPU.wv,sizeof(float)*gpuProduct.nVertices,
			      hipMemcpyDeviceToHost, stream));
    gpuProduct.chi2.resize(gpuProduct.nVertices);
    cudaCheck(hipMemcpyAsync(gpuProduct.chi2.data(),onGPU.chi2,sizeof(float)*gpuProduct.nVertices,
			      hipMemcpyDeviceToHost, stream));
    
    hipStreamSynchronize(stream);
    
    return gpuProduct;
  }

  
  
} // end namespace

