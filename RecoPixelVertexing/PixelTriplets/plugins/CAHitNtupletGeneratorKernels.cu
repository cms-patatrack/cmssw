#include "RecoPixelVertexing/PixelTriplets/plugins/CAHitNtupletGeneratorKernelsImpl.h"


template
void CAHitNtupletGeneratorKernelsGPU::fillHitDetIndices(HitsView const *hv, TkSoA *tracks_d, hipStream_t cudaStream);

template
void CAHitNtupletGeneratorKernelsGPU::launchKernels(HitsOnCPU const &hh, TkSoA *tracks_d, hipStream_t cudaStream);

template
void CAHitNtupletGeneratorKernelsGPU::buildDoublets(HitsOnCPU const &hh, hipStream_t stream);

template
void CAHitNtupletGeneratorKernelsGPU::classifyTuples(HitsOnCPU const &hh, TkSoA *tracks_d, hipStream_t cudaStream);

template
void CAHitNtupletGeneratorKernelsGPU::printCounters(Counters const *counters);
