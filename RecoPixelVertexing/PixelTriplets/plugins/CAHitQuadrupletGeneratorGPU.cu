#include "hip/hip_runtime.h"
//
// Author: Felice Pantaleo, CERN
//

#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "CAHitQuadrupletGeneratorGPU.h"
#include "GPUCACell.h"
#include "gpuPixelDoublets.h"
#include <cstdint>

using HitsOnCPU = siPixelRecHitsHeterogeneousProduct::HitsOnCPU;

__global__ void
kernel_checkOverflows(GPU::SimpleVector<Quadruplet> *foundNtuplets,
               GPUCACell *cells, uint32_t const * nCells,
               GPU::VecArray< unsigned int, 256> *isOuterHitOfCell,
               uint32_t nHits) {

 auto idx = threadIdx.x + blockIdx.x * blockDim.x;
 #ifdef GPU_DEBUG
 if (0==idx)
   printf("number of found cells %d\n",*nCells);
 #endif
 if (idx < (*nCells) ) {
   auto &thisCell = cells[idx];
   if (thisCell.theOuterNeighbors.full()) //++tooManyNeighbors[thisCell.theLayerPairId];
     printf("OuterNeighbors overflow %d in %d\n",idx,thisCell.theLayerPairId);
 }
 if (idx < nHits) {
   if (isOuterHitOfCell[idx].full()) // ++tooManyOuterHitOfCell;
     printf("OuterHitOfCell overflow %d\n", idx); 
 }

}


__global__ void
kernel_connect(GPU::SimpleVector<Quadruplet> *foundNtuplets,
               GPUCACell *cells, uint32_t const * nCells,
               GPU::VecArray< unsigned int, 256> *isOuterHitOfCell,
               float ptmin, 
               float region_origin_radius, const float thetaCut,
               const float phiCut, const float hardPtCut,
               unsigned int maxNumberOfDoublets_, unsigned int maxNumberOfHits_) {

  float region_origin_x =0.;
  float region_origin_y =0.;

  auto cellIndex = threadIdx.x + blockIdx.x * blockDim.x;

  if (0==cellIndex) foundNtuplets->reset(); // ready for next kernel

  if (cellIndex >= (*nCells) ) return;
  auto &thisCell = cells[cellIndex];
  auto innerHitId = thisCell.get_inner_hit_id();
  auto numberOfPossibleNeighbors = isOuterHitOfCell[innerHitId].size();
  for (auto j = 0; j < numberOfPossibleNeighbors; ++j) {
     auto otherCell = isOuterHitOfCell[innerHitId][j];

     if (thisCell.check_alignment_and_tag(
                 cells, otherCell, ptmin, region_origin_x, region_origin_y,
                  region_origin_radius, thetaCut, phiCut, hardPtCut)
        ) {
          cells[otherCell].theOuterNeighbors.push_back(cellIndex);
     }
  }
}

__global__ void kernel_find_ntuplets(
    GPUCACell *cells, uint32_t const * nCells,
    GPU::SimpleVector<Quadruplet> *foundNtuplets,
    unsigned int minHitsPerNtuplet,
    unsigned int maxNumberOfDoublets_)
{

  auto cellIndex = threadIdx.x + blockIdx.x * blockDim.x;
  if (cellIndex >= (*nCells) ) return;
  auto &thisCell = cells[cellIndex];
  if (thisCell.theLayerPairId!=0 && thisCell.theLayerPairId!=3 && thisCell.theLayerPairId!=8) return; // inner layer is 0 FIXME
  GPU::VecArray<unsigned int, 3> stack;
  stack.reset();
  thisCell.find_ntuplets(cells, foundNtuplets, stack, minHitsPerNtuplet);
  assert(stack.size()==0);
  // printf("in %d found quadruplets: %d\n", cellIndex, foundNtuplets->size());
}

__global__ void
kernel_print_found_ntuplets(GPU::SimpleVector<Quadruplet> *foundNtuplets, int maxPrint) {
  for (int i = 0; i < std::min(maxPrint,foundNtuplets->size()); ++i) {
    printf("\nquadruplet %d: %d %d %d %d\n", i,
           (*foundNtuplets)[i].hitId[0],
           (*foundNtuplets)[i].hitId[1],
           (*foundNtuplets)[i].hitId[2],
           (*foundNtuplets)[i].hitId[3]
          );
         
  }
}

void CAHitQuadrupletGeneratorGPU::deallocateOnGPU()
{
  for (size_t i = 0; i < h_foundNtupletsVec_.size(); ++i)
  {
    hipHostFree(h_foundNtupletsVec_[i]);
    hipHostFree(h_foundNtupletsData_[i]);
    hipFree(d_foundNtupletsVec_[i]);
    hipFree(d_foundNtupletsData_[i]);
  }

  hipFree(device_theCells_);
  hipFree(device_isOuterHitOfCell_);
  hipFree(device_nCells_);
}

void CAHitQuadrupletGeneratorGPU::allocateOnGPU()
{
  //////////////////////////////////////////////////////////
  // ALLOCATIONS FOR THE INTERMEDIATE RESULTS (STAYS ON WORKER)
  //////////////////////////////////////////////////////////

  cudaCheck(hipMalloc(&device_theCells_,
             maxNumberOfLayerPairs_ * maxNumberOfDoublets_ * sizeof(GPUCACell)));
  cudaCheck(hipMalloc(&device_nCells_, sizeof(uint32_t)));
  cudaCheck(hipMemset(device_nCells_, 0, sizeof(uint32_t)));

  cudaCheck(hipMalloc(&device_isOuterHitOfCell_,
             maxNumberOfLayers_ * maxNumberOfHits_ * sizeof(GPU::VecArray<unsigned int, maxCellsPerHit_>)));
  cudaCheck(hipMemset(device_isOuterHitOfCell_, 0,
             maxNumberOfLayers_ * maxNumberOfHits_ * sizeof(GPU::VecArray<unsigned int, maxCellsPerHit_>)));

  h_foundNtupletsVec_.resize(maxNumberOfRegions_);
  h_foundNtupletsData_.resize(maxNumberOfRegions_);
  d_foundNtupletsVec_.resize(maxNumberOfRegions_);
  d_foundNtupletsData_.resize(maxNumberOfRegions_);

  // FIXME this could be rewritten with a single pair of hipHostMalloc / hipMalloc
  for (int i = 0; i < maxNumberOfRegions_; ++i) {
    cudaCheck(hipHostMalloc(&h_foundNtupletsData_[i],  sizeof(Quadruplet) * maxNumberOfQuadruplets_));
    cudaCheck(hipHostMalloc(&h_foundNtupletsVec_[i],   sizeof(GPU::SimpleVector<Quadruplet>)));
    new(h_foundNtupletsVec_[i]) GPU::SimpleVector<Quadruplet>(maxNumberOfQuadruplets_, h_foundNtupletsData_[i]);
    cudaCheck(hipMalloc(&d_foundNtupletsData_[i],      sizeof(Quadruplet) * maxNumberOfQuadruplets_));
    cudaCheck(hipMemset(d_foundNtupletsData_[i], 0x00, sizeof(Quadruplet) * maxNumberOfQuadruplets_));
    cudaCheck(hipMalloc(&d_foundNtupletsVec_[i],       sizeof(GPU::SimpleVector<Quadruplet>)));
    GPU::SimpleVector<Quadruplet> tmp_foundNtuplets(maxNumberOfQuadruplets_, d_foundNtupletsData_[i]);
    cudaCheck(hipMemcpy(d_foundNtupletsVec_[i], & tmp_foundNtuplets, sizeof(GPU::SimpleVector<Quadruplet>), hipMemcpyDefault));
  }

}

void CAHitQuadrupletGeneratorGPU::launchKernels(const TrackingRegion &region,
                                                int regionIndex, HitsOnCPU const & hh,
                                                bool transferToCPU,
                                                hipStream_t cudaStream)
{
  assert(regionIndex < maxNumberOfRegions_);
  assert(0==regionIndex);

  h_foundNtupletsVec_[regionIndex]->reset();

  auto nhits = hh.nHits;

  auto numberOfBlocks = (maxNumberOfDoublets_ + 512 - 1)/512;
  kernel_connect<<<numberOfBlocks, 512, 0, cudaStream>>>(
      d_foundNtupletsVec_[regionIndex], // needed only to be reset, ready for next kernel
      device_theCells_, device_nCells_,
      device_isOuterHitOfCell_,
      region.ptMin(), 
      region.originRBound(), caThetaCut, caPhiCut, caHardPtCut,
      maxNumberOfDoublets_, maxNumberOfHits_
  );
  cudaCheck(hipGetLastError());

  kernel_find_ntuplets<<<numberOfBlocks, 512, 0, cudaStream>>>(
      device_theCells_, device_nCells_,
      d_foundNtupletsVec_[regionIndex],
      4, maxNumberOfDoublets_);
  cudaCheck(hipGetLastError());


  numberOfBlocks = (std::max(int(nhits),maxNumberOfDoublets_) + 512 - 1)/512;
  kernel_checkOverflows<<<numberOfBlocks, 512, 0, cudaStream>>>(
                        d_foundNtupletsVec_[regionIndex],
                        device_theCells_, device_nCells_,
                        device_isOuterHitOfCell_, nhits
                       );


  // kernel_print_found_ntuplets<<<1,1,0, cudaStream>>>(d_foundNtupletsVec_[regionIndex],10);

  if(transferToCPU) {
    cudaCheck(hipMemcpyAsync(h_foundNtupletsVec_[regionIndex], d_foundNtupletsVec_[regionIndex],
                              sizeof(GPU::SimpleVector<Quadruplet>),
                              hipMemcpyDeviceToHost, cudaStream));

    cudaCheck(hipMemcpyAsync(h_foundNtupletsData_[regionIndex], d_foundNtupletsData_[regionIndex],
                              maxNumberOfQuadruplets_*sizeof(Quadruplet),
                              hipMemcpyDeviceToHost, cudaStream));
  }
}

void CAHitQuadrupletGeneratorGPU::cleanup(hipStream_t cudaStream) {
  // this lazily resets temporary memory for the next event, and is not needed for reading the output
  cudaCheck(hipMemsetAsync(device_isOuterHitOfCell_, 0,
                            maxNumberOfLayers_ * maxNumberOfHits_ * sizeof(GPU::VecArray<unsigned int, maxCellsPerHit_>),
                            cudaStream));
  cudaCheck(hipMemsetAsync(device_nCells_,0,sizeof(uint32_t),cudaStream));
}

std::vector<std::array<int, 4>>
CAHitQuadrupletGeneratorGPU::fetchKernelResult(int regionIndex)
{
  assert(0==regionIndex);
  h_foundNtupletsVec_[regionIndex]->set_data(h_foundNtupletsData_[regionIndex]);

  std::vector<std::array<int, 4>> quadsInterface(h_foundNtupletsVec_[regionIndex]->size());
  for (int i = 0; i < h_foundNtupletsVec_[regionIndex]->size(); ++i) {
    for (int j = 0; j<4; ++j) quadsInterface[i][j] = (*h_foundNtupletsVec_[regionIndex])[i].hitId[j];
  }
  return quadsInterface;
}

void CAHitQuadrupletGeneratorGPU::buildDoublets(HitsOnCPU const & hh, hipStream_t stream) {
   auto nhits = hh.nHits;

  int threadsPerBlock = 256;
  int blocks = (3*nhits + threadsPerBlock - 1) / threadsPerBlock;
  gpuPixelDoublets::getDoubletsFromHisto<<<blocks, threadsPerBlock, 0, stream>>>(device_theCells_,device_nCells_,hh.gpu_d, device_isOuterHitOfCell_);
  cudaCheck(hipGetLastError());
}
