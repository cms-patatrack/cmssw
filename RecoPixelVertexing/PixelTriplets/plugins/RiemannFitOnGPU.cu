#include "RiemannFitOnGPU.h"

template
void HelixFitOnGPU::launchRiemannKernels<cudaCompat::GPUTraits>(HitsView const *hv,
                                         uint32_t nhits,
                                         uint32_t maxNumberOfTuples,
                                         hipStream_t stream);
