#include "hip/hip_runtime.h"
#include <Eigen/Core>
#include <Eigen/Eigenvalues>

#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "PixelTrackReconstructionGPU.h"

using namespace Eigen;

__global__ void
KernelFastFitAllHits(float *hits_and_covariances,
    int hits_in_fit,
    int cumulative_size,
    float B,
    Rfit::helix_fit *results,
    Rfit::Matrix3xNd *hits,
    Rfit::Matrix3Nd *hits_cov,
    Rfit::circle_fit *circle_fit,
    Vector4d *fast_fit,
    Rfit::line_fit *line_fit)
{
  // Reshape Eigen components from hits_and_covariances, using proper thread and block indices
  // Perform the fit
  // Store the results in the proper vector, using again correct indices

  // Loop for hits_in_fit times:
  //   first 3 are the points
  //   the rest is the covariance matrix, 3x3
  int start = (blockIdx.x * blockDim.x + threadIdx.x) * hits_in_fit * 12;
  int helix_start = (blockIdx.x * blockDim.x + threadIdx.x);
  if (start >= cumulative_size) {
    return;
  }

#ifdef GPU_DEBUG
  printf("BlockDim.x: %d, BlockIdx.x: %d, threadIdx.x: %d, start: %d, cumulative_size: %d\n",
      blockDim.x, blockIdx.x, threadIdx.x, start, cumulative_size);
#endif

  hits[helix_start].resize(3, hits_in_fit);
  hits_cov[helix_start].resize(3 * hits_in_fit, 3 * hits_in_fit);

  // Prepare data structure (stack)
  for (unsigned int i = 0; i < hits_in_fit; ++i) {
    hits[helix_start].col(i) << hits_and_covariances[start],
        hits_and_covariances[start + 1], hits_and_covariances[start + 2];
    start += 3;

    for (auto j = 0; j < 3; ++j) {
      for (auto l = 0; l < 3; ++l) {
        hits_cov[helix_start](i + j * hits_in_fit, i + l * hits_in_fit) =
            hits_and_covariances[start];
        start++;
      }
    }
  }

  Rfit::Fast_fit(hits[helix_start], fast_fit[helix_start]);
}

__global__
void kernelComputeCircleWeights(int hits_in_fit, int cumulative_size, float B,
    Rfit::Matrix3xNd *hits,
    Rfit::Matrix3Nd *hits_cov,
    Vector4d *fast_fit,
    Rfit::covariancesForCircle * cov)
{
  int start = (blockIdx.x * blockDim.x + threadIdx.x) * hits_in_fit * 12;
  int helix_start = (blockIdx.x * blockDim.x + threadIdx.x);
  if (start >= cumulative_size) {
    return;
  }

#ifdef GPU_DEBUG
  printf("blockDim.x: %d, blockIdx.x: %d, threadIdx.x: %d, helix_start: %d, cumulative_size: %d\n",
         blockDim.x, blockIdx.x, threadIdx.x, helix_start, cumulative_size);
#endif
  auto n = hits[helix_start].cols();

  Rfit::VectorNd rad = (hits[helix_start].block(0, 0, 2, n).colwise().norm());

  cov[helix_start].V.resize(2 * n, 2 * n);
  cov[helix_start].V = hits_cov[helix_start].block(0, 0, 2 * n, 2 * n);
  Rfit::printIt(&cov[helix_start].V, "kernelComputeCircleWeights: ");
  Rfit::ComputeCircleWeights(hits[helix_start].block(0, 0, 2, n),
      fast_fit[helix_start], rad, B, cov[helix_start]);
}

__global__
void kernelCircleFitComputeCircleParametersAndErrors(
    int hits_in_fit,
    int cumulative_size,
    Rfit::Matrix3xNd *hits,
    Rfit::circle_fit *circle_fit,
    Rfit::covariancesForCircle * cov)
{
  int start = (blockIdx.x * blockDim.x + threadIdx.x) * hits_in_fit * 12;
  int helix_start = (blockIdx.x * blockDim.x + threadIdx.x);
  if (start >= cumulative_size) {
    return;
  }

#ifdef GPU_DEBUG
  printf("blockDim.x: %d, blockIdx.x: %d, threadIdx.x: %d, helix_start: %d, cumulative_size: %d\n",
         blockDim.x, blockIdx.x, threadIdx.x, helix_start, foundNtuplets->size());
#endif
  auto n = hits[helix_start].cols();

  Rfit::ComputeCircleParametersAndErrors(hits[helix_start].block(0, 0, 2, n),
                                         cov[helix_start],
                                         circle_fit[helix_start]);

}

__global__ void
__launch_bounds__(256, 2)
KernelCircleFitAllHits(int hits_in_fit,
                       int cumulative_size, float B, Rfit::helix_fit *results,
                       Rfit::Matrix3xNd *hits, Rfit::Matrix3Nd *hits_cov,
                       Rfit::circle_fit *circle_fit, Vector4d *fast_fit,
                       Rfit::line_fit *line_fit, Rfit::covariancesForCircle * cov)// Rfit::ArrayNd * vcs, Rfit::MatrixNd * C, Rfit::MatrixNd * D)
{
  // Reshape Eigen components from hits_and_covariances, using proper thread and block indices
  // Perform the fit
  // Store the results in the proper vector, using again correct indices

  // Loop for hits_in_fit times:
  //   first 3 are the points
  //   the rest is the covariance matrix, 3x3
  int start = (blockIdx.x * blockDim.x + threadIdx.x) * hits_in_fit * 12;
  int helix_start = (blockIdx.x * blockDim.x + threadIdx.x);
  if (start >= cumulative_size) {
    return;
  }

#ifdef GPU_DEBUG
    printf("BlockDim.x: %d, BlockIdx.x: %d, threadIdx.x: %d, start: %d, "
           "cumulative_size: %d\n",
           blockDim.x, blockIdx.x, threadIdx.x, start, cumulative_size);
#endif
  u_int n = hits[helix_start].cols();

  Rfit::VectorNd rad = (hits[helix_start].block(0, 0, 2, n).colwise().norm());

  Rfit::Circle_fit(hits[helix_start].block(0, 0, 2, n),
                   hits_cov[helix_start].block(0, 0, 2 * n, 2 * n),
                   fast_fit[helix_start], rad, B,
                   cov[helix_start],
                   circle_fit[helix_start], true);

#ifdef GPU_DEBUG
    printf("KernelCircleFitAllHits circle.par(0): %d %f\n", helix_start,
           circle_fit[helix_start].par(0));
    printf("KernelCircleFitAllHits circle.par(1): %d %f\n", helix_start,
           circle_fit[helix_start].par(1));
    printf("KernelCircleFitAllHits circle.par(2): %d %f\n", helix_start,
           circle_fit[helix_start].par(2));
#endif

}

__global__ void
KernelLineFitAllHits(int hits_in_fit,
                     int cumulative_size, float B, Rfit::helix_fit *results,
                     Rfit::Matrix3xNd *hits, Rfit::Matrix3Nd *hits_cov,
                     Rfit::circle_fit *circle_fit, Vector4d *fast_fit,
                     Rfit::line_fit *line_fit)
{
  // Reshape Eigen components from hits_and_covariances, using proper thread and block indices
  // Perform the fit
  // Store the results in the proper vector, using again correct indices

  // Loop for hits_in_fit times:
  //   first 3 are the points
  //   the rest is the covariance matrix, 3x3
  int start = (blockIdx.x * blockDim.x + threadIdx.x) * hits_in_fit * 12;
  int helix_start = (blockIdx.x * blockDim.x + threadIdx.x);
  if (start >= cumulative_size) {
    return;
  }

#ifdef GPU_DEBUG

  printf("BlockDim.x: %d, BlockIdx.x: %d, threadIdx.x: %d, start: %d, "
      "cumulative_size: %d\n",
      blockDim.x, blockIdx.x, threadIdx.x, start, cumulative_size);
#endif

  Rfit::Line_fit(hits[helix_start], hits_cov[helix_start],
                 circle_fit[helix_start], fast_fit[helix_start], B, line_fit[helix_start], true);

  transformCircleParamsToTrackRepresentation(circle_fit[helix_start], B, true);

  // Grab helix_fit from the proper location in the output vector
  Rfit::helix_fit &helix = results[helix_start];
  helix.par << circle_fit[helix_start].par, line_fit[helix_start].par;

  // TODO: pass properly error booleans

  helix.cov = MatrixXd::Zero(5, 5);
  helix.cov.block(0, 0, 3, 3) = circle_fit[helix_start].cov;
  helix.cov.block(3, 3, 2, 2) = line_fit[helix_start].cov;

  helix.q = circle_fit[helix_start].q;
  helix.chi2_circle = circle_fit[helix_start].chi2;
  helix.chi2_line = line_fit[helix_start].chi2;

#ifdef GPU_DEBUG

    printf("KernelLineFitAllHits line.par(0): %d %f\n", helix_start,
           circle_fit[helix_start].par(0));
    printf("KernelLineFitAllHits line.par(1): %d %f\n", helix_start,
           line_fit[helix_start].par(1));
#endif
}

void PixelTrackReconstructionGPU::launchKernelFit(
    float *hits_and_covariancesGPU, int cumulative_size, int hits_in_fit,
    float B, Rfit::helix_fit *results)
{
  const dim3 threads_per_block(256, 1);
  int num_blocks = cumulative_size / (hits_in_fit * 12) / threads_per_block.x + 1;
  auto numberOfSeeds = cumulative_size / (hits_in_fit * 12);

  Rfit::Matrix3xNd *hitsGPU;
  cudaCheck(hipMalloc(&hitsGPU, numberOfSeeds * sizeof(Rfit::Matrix3xNd(3, 4))));
  cudaCheck(hipMemset(hitsGPU, 0x00, numberOfSeeds * sizeof(Rfit::Matrix3xNd(3, 4))));

  Rfit::Matrix3Nd *hits_covGPU = nullptr;
  cudaCheck(hipMalloc(&hits_covGPU, numberOfSeeds * sizeof(Rfit::Matrix3Nd(12, 12))));
  cudaCheck(hipMemset(hits_covGPU, 0x00, numberOfSeeds * sizeof(Rfit::Matrix3Nd(12, 12))));

  Vector4d *fast_fit_resultsGPU = nullptr;
  cudaCheck(hipMalloc(&fast_fit_resultsGPU, numberOfSeeds * sizeof(Vector4d)));
  cudaCheck(hipMemset(fast_fit_resultsGPU, 0x00, numberOfSeeds * sizeof(Vector4d)));

  Rfit::circle_fit *circle_fit_resultsGPU = nullptr;
  cudaCheck(hipMalloc(&circle_fit_resultsGPU, numberOfSeeds * sizeof(Rfit::circle_fit)));
  cudaCheck(hipMemset(circle_fit_resultsGPU, 0x00, numberOfSeeds * sizeof(Rfit::circle_fit)));

  Rfit::line_fit *line_fit_resultsGPU = nullptr;
  cudaCheck(hipMalloc(&line_fit_resultsGPU, numberOfSeeds * sizeof(Rfit::line_fit)));
  cudaCheck(hipMemset(line_fit_resultsGPU, 0x00, numberOfSeeds * sizeof(Rfit::line_fit)));

  Rfit::covariancesForCircle * cov_for_circle = nullptr;
  cudaCheck(hipMalloc(&cov_for_circle, sizeof(Rfit::covariancesForCircle)*numberOfSeeds));
  cudaCheck(hipMemset(cov_for_circle, 0x00, sizeof(Rfit::covariancesForCircle)*numberOfSeeds));

  KernelFastFitAllHits<<<num_blocks, threads_per_block>>>(
      hits_and_covariancesGPU, hits_in_fit, cumulative_size, B, results,
      hitsGPU, hits_covGPU, circle_fit_resultsGPU, fast_fit_resultsGPU,
      line_fit_resultsGPU);
  cudaCheck(hipGetLastError());

  kernelComputeCircleWeights<<<num_blocks, threads_per_block>>>(
      hits_in_fit, cumulative_size, B,
      hitsGPU, hits_covGPU, fast_fit_resultsGPU, cov_for_circle);
  cudaCheck(hipGetLastError());

  KernelCircleFitAllHits<<<num_blocks, threads_per_block>>>(
      hits_in_fit, cumulative_size, B, results,
      hitsGPU, hits_covGPU, circle_fit_resultsGPU, fast_fit_resultsGPU,
      line_fit_resultsGPU, cov_for_circle);
  cudaCheck(hipGetLastError());

  kernelCircleFitComputeCircleParametersAndErrors<<<num_blocks, threads_per_block>>>(
      hits_in_fit, cumulative_size, hitsGPU, circle_fit_resultsGPU, cov_for_circle);
  cudaCheck(hipGetLastError());

  KernelLineFitAllHits<<<num_blocks, threads_per_block>>>(
      hits_in_fit, cumulative_size, B, results,
      hitsGPU, hits_covGPU, circle_fit_resultsGPU, fast_fit_resultsGPU,
      line_fit_resultsGPU);
  cudaCheck(hipGetLastError());

  hipFree(hitsGPU);
  hipFree(hits_covGPU);
  hipFree(fast_fit_resultsGPU);
  hipFree(circle_fit_resultsGPU);
  hipFree(line_fit_resultsGPU);
  hipFree(cov_for_circle);
}
