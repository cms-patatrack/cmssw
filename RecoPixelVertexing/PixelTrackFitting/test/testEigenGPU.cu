#include "hip/hip_runtime.h"
#include <iostream>

#include <Eigen/Core>
#include <Eigen/Eigenvalues>

using namespace Eigen;

__host__ __device__ void eigenValues(Matrix3d * m, Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType * ret) {
  printf("Matrix(0,0): %f\n", (*m)(0,0));
  printf("Matrix(1,1): %f\n", (*m)(1,1));
  printf("Matrix(2,2): %f\n", (*m)(2,2));
  SelfAdjointEigenSolver<Matrix3d> es;
  es.computeDirect(*m);
  (*ret) = es.eigenvalues();
  return;
}

__global__ void kernel(Matrix3d * m, Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType * ret) {
  eigenValues(m, ret);
}

bool isEqualFuzzy(double a, double b) {
  constexpr double epsilon = 1e-6;
  return std::abs(a-b) < std::min(std::abs(a), std::abs(b))*epsilon;
}

int main (int argc, char * argv[]) {
  
  Matrix3d m = Matrix3d::Random();
  Matrix3d mt = m.transpose();
  m += mt;
  Matrix3d * m_gpu;
  Matrix3d * mgpudebug = new Matrix3d();
  Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType *ret = new Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType;
  Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType *ret1 = new Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType;
  Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType *ret_gpu;
  eigenValues(&m, ret);
  std::cout << "Generated Matrix M 3x3:\n" << m << std::endl;
  std::cout << "The eigenvalues of M are:" << std::endl << (*ret) << std::endl;
  std::cout << "*************************\n\n" << std::endl;

  hipMalloc((void **)&m_gpu, sizeof(Matrix3d));
  hipMalloc((void **)&ret_gpu, sizeof(Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType));
  hipMemcpy(m_gpu, &m, sizeof(Matrix3d), hipMemcpyHostToDevice);

  kernel<<<1,1>>>(m_gpu, ret_gpu);

  hipDeviceSynchronize();

  hipMemcpy(mgpudebug, m_gpu, sizeof(Matrix3d), hipMemcpyDeviceToHost);
  hipMemcpy(ret1, ret_gpu, sizeof(Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType), hipMemcpyDeviceToHost);
  std::cout << "GPU Generated Matrix M 3x3:\n" << (*mgpudebug) << std::endl;
  std::cout << "GPU The eigenvalues of M are:" << std::endl << (*ret1) << std::endl;
  std::cout << "*************************\n\n" << std::endl;

  std::cout << "Ratio: " << (*ret)(0,0)/(*ret1)(0,0) << std::endl;
  std::cout << "Ratio: " << (*ret)(1,0)/(*ret1)(1,0) << std::endl;
  std::cout << "Ratio: " << (*ret)(2,0)/(*ret1)(2,0) << std::endl;
  assert(isEqualFuzzy((*ret)(0,0), (*ret1)(0,0)));
  assert(isEqualFuzzy((*ret)(1,0), (*ret1)(1,0)));
  assert(isEqualFuzzy((*ret)(2,0), (*ret1)(2,0)));
  return 0;
}
