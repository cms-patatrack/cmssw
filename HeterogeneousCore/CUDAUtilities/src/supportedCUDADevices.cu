#include "hip/hip_runtime.h"
#include <vector>

#include <hip/hip_runtime.h>

#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "HeterogeneousCore/CUDAUtilities/interface/supportedCUDADevices.h"

__global__
void isSupported(bool * result) {
  * result = true;
}

std::vector<int> supportedCUDADevices() {
  int devices = 0;
  auto status = hipGetDeviceCount(&devices);
  if (status != hipSuccess or devices == 0) {
    return {};
  }

  std::vector<int> supportedDevices;
  supportedDevices.reserve(devices);

  for (int i = 0; i < devices; ++i) {
    cudaCheck(hipSetDevice(i));
    bool supported = false;
    bool * supported_d;
    cudaCheck(hipMalloc(&supported_d, sizeof(bool)));
    cudaCheck(hipMemset(supported_d, 0x00, sizeof(bool)));
    isSupported<<<1,1>>>(supported_d);
    // swallow any eventual error from launching the kernel on an unsupported device
    hipGetLastError();
    cudaCheck(hipDeviceSynchronize());
    cudaCheck(hipMemcpy(& supported, supported_d, sizeof(bool), hipMemcpyDeviceToHost));
    cudaCheck(hipFree(supported_d));
    if (supported) {
      supportedDevices.push_back(i);
    }
    cudaCheck(hipDeviceReset());
  }

  return supportedDevices;
}
