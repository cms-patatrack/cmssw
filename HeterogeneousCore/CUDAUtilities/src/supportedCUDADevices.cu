#include "hip/hip_runtime.h"
#include <map>

#include <hip/hip_runtime.h>

#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "HeterogeneousCore/CUDAUtilities/interface/supportedCUDADevices.h"

__global__
void isSupported(bool * result) {
  * result = true;
}

std::map<int, std::pair<int, int>> supportedCUDADevices(bool reset) {
  std::map<int, std::pair<int, int>> capabilities;

  int devices = 0;
  auto status = hipGetDeviceCount(&devices);
  if (hipSuccess != status) {
    return capabilities;
  }

  for (int i = 0; i < devices; ++i) {
    cudaCheck(hipSetDevice(i));
    bool supported = false;
    bool * supported_d;
    cudaCheck(hipMalloc(&supported_d, sizeof(bool)));
    cudaCheck(hipMemset(supported_d, 0x00, sizeof(bool)));
    isSupported<<<1,1>>>(supported_d);
    // swallow any eventual error from launching the kernel on an unsupported device
    hipGetLastError();
    cudaCheck(hipDeviceSynchronize());
    cudaCheck(hipMemcpy(& supported, supported_d, sizeof(bool), hipMemcpyDeviceToHost));
    cudaCheck(hipFree(supported_d));
    if (supported) {
      hipDeviceProp_t properties;
      cudaCheck(hipGetDeviceProperties(&properties, i));
      capabilities[i] = std::make_pair(properties.major, properties.minor);
    }
    if (reset) {
      cudaCheck(hipDeviceReset());
    }
  }

  return capabilities;
}
