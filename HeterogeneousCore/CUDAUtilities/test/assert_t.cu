#include "hip/hip_runtime.h"
#include "HeterogeneousCore/CUDAUtilities/interface/cuda_assert.h"
#include "HeterogeneousCore/CUDAUtilities/interface/exitSansCUDADevices.h"

__global__
void testIt(int c){
  assert(c==1);
}

int main(int c, char **) {
  exitSansCUDADevices();

  testIt<<<1,1>>>(c);
  hipDeviceSynchronize();
  return c==1;
}
